#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

/** @file   testbed_nerf.cu
 *  @author Thomas Müller & Alex Evans, NVIDIA
 */

#include <neural-graphics-primitives/adam_optimizer.h>
#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/envmap.cuh>
#include <neural-graphics-primitives/nerf_loader.h>
#include <neural-graphics-primitives/nerf_network.h>
#include <neural-graphics-primitives/marching_cubes.h>
#include <neural-graphics-primitives/render_buffer.h>
#include <neural-graphics-primitives/testbed.h>
#include <neural-graphics-primitives/trainable_buffer.cuh>

#include <tiny-cuda-nn/encodings/grid.h>
#include <tiny-cuda-nn/loss.h>
#include <tiny-cuda-nn/network.h>
#include <tiny-cuda-nn/network_with_input_encoding.h>
#include <tiny-cuda-nn/optimizer.h>
#include <tiny-cuda-nn/trainer.h>

#include <filesystem/directory.h>
#include <filesystem/path.h>

#ifdef copysign
#undef copysign
#endif

using namespace Eigen;
using namespace tcnn;
namespace fs = filesystem;

NGP_NAMESPACE_BEGIN

inline constexpr __device__ float NERF_RENDERING_NEAR_DISTANCE() { return 0.05f; }
inline constexpr __device__ uint32_t NERF_STEPS() { return 1024; } // finest number of steps per unit length
inline constexpr __device__ uint32_t NERF_CASCADES() { return 5; }

inline constexpr __device__ float SQRT3() { return 1.73205080757f; }
inline constexpr __device__ float STEPSIZE() { return (SQRT3() / NERF_STEPS()); } // for nerf raymarch
inline constexpr __device__ float MIN_CONE_STEPSIZE() { return STEPSIZE(); }
// Maximum step size is the width of the coarsest gridsize cell.
inline constexpr __device__ float MAX_CONE_STEPSIZE() { return STEPSIZE() * (1<<(NERF_CASCADES()-1)) * NERF_STEPS() / NERF_GRIDSIZE(); }

// Used to index into the PRNG stream. Must be larger than the number of
// samples consumed by any given training ray.
inline constexpr __device__ uint32_t N_MAX_RANDOM_SAMPLES_PER_RAY() { return 8; }

// Any alpha below this is considered "invisible" and is thus culled away.
inline constexpr __device__ float NERF_MIN_OPTICAL_THICKNESS() { return 0.01f; }

static constexpr uint32_t MARCH_ITER = 10000;

static constexpr uint32_t MIN_STEPS_INBETWEEN_COMPACTION = 1;
static constexpr uint32_t MAX_STEPS_INBETWEEN_COMPACTION = 8;


inline __host__ __device__ uint32_t grid_mip_offset(uint32_t mip) {
	return (NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_GRIDSIZE()) * mip;
}

inline __host__ __device__ float calc_cone_angle(float cosine, const Eigen::Vector2f& focal_length, float cone_angle_constant) {
	// Pixel size. Doesn't always yield a good performance vs. quality
	// trade off. Especially if training pixels have a much different
	// size than rendering pixels.
	// return cosine*cosine / focal_length.mean();

	return cone_angle_constant;
}

inline __host__ __device__ float calc_dt(float t, float cone_angle) {
	return tcnn::clamp(t*cone_angle, MIN_CONE_STEPSIZE(), MAX_CONE_STEPSIZE());
}

struct LossAndGradient {
	Eigen::Array3f loss;
	Eigen::Array3f gradient;

	__host__ __device__ LossAndGradient operator*(float scalar) {
		return {loss * scalar, gradient * scalar};
	}

	__host__ __device__ LossAndGradient operator/(float scalar) {
		return {loss / scalar, gradient / scalar};
	}
};

inline __device__ Array3f copysign(const Array3f& a, const Array3f& b) {
	return {
		copysignf(a.x(), b.x()),
		copysignf(a.y(), b.y()),
		copysignf(a.z(), b.z()),
	};
}

inline __device__ LossAndGradient l2_loss(const Array3f& target, const Array3f& prediction) {
	Array3f difference = prediction - target;
	return {
		difference * difference,
		2.0f * difference
	};
}

inline __device__ LossAndGradient relative_l2_loss(const Array3f& target, const Array3f& prediction) {
	Array3f difference = prediction - target;
	Array3f factor = (prediction * prediction + Array3f::Constant(1e-2f)).inverse();
	return {
		difference * difference * factor,
		2.0f * difference * factor
	};
}

inline __device__ LossAndGradient l1_loss(const Array3f& target, const Array3f& prediction) {
	Array3f difference = prediction - target;
	return {
		difference.abs(),
		copysign(Array3f::Ones(), difference),
	};
}

inline __device__ LossAndGradient huber_loss(const Array3f& target, const Array3f& prediction, float alpha = 1) {
	Array3f difference = prediction - target;
	Array3f abs_diff = difference.abs();
	Array3f square = 0.5f/alpha * difference * difference;
	return {
		{
			abs_diff.x() > alpha ? (abs_diff.x() - 0.5f * alpha) : square.x(),
			abs_diff.y() > alpha ? (abs_diff.y() - 0.5f * alpha) : square.y(),
			abs_diff.z() > alpha ? (abs_diff.z() - 0.5f * alpha) : square.z(),
		},
		{
			abs_diff.x() > alpha ? (difference.x() > 0 ? 1.0f : -1.0f) : (difference.x() / alpha),
			abs_diff.y() > alpha ? (difference.y() > 0 ? 1.0f : -1.0f) : (difference.y() / alpha),
			abs_diff.z() > alpha ? (difference.z() > 0 ? 1.0f : -1.0f) : (difference.z() / alpha),
		},
	};
}

inline __device__ LossAndGradient log_l1_loss(const Array3f& target, const Array3f& prediction) {
	Array3f difference = prediction - target;
	Array3f divisor = difference.abs() + Array3f::Ones();
	return {
		divisor.log(),
		copysign(divisor.inverse(), difference),
	};
}

inline __device__ LossAndGradient smape_loss(const Array3f& target, const Array3f& prediction) {
	Array3f difference = prediction - target;
	Array3f factor = (0.5f * (prediction.abs() + target.abs()) + Array3f::Constant(1e-2f)).inverse();
	return {
		difference.abs() * factor,
		copysign(factor, difference),
	};
}

inline __device__ LossAndGradient mape_loss(const Array3f& target, const Array3f& prediction) {
	Array3f difference = prediction - target;
	Array3f factor = (prediction.abs() + Array3f::Constant(1e-2f)).inverse();
	return {
		difference.abs() * factor,
		copysign(factor, difference),
	};
}

inline __device__ float distance_to_next_voxel(const Vector3f& pos, const Vector3f& dir, const Vector3f& idir, uint32_t res) { // dda like step
	Vector3f p = res * pos;
	float tx = (floorf(p.x() + 0.5f + 0.5f * sign(dir.x())) - p.x()) * idir.x();
	float ty = (floorf(p.y() + 0.5f + 0.5f * sign(dir.y())) - p.y()) * idir.y();
	float tz = (floorf(p.z() + 0.5f + 0.5f * sign(dir.z())) - p.z()) * idir.z();
	float t = min(min(tx, ty), tz);

	return fmaxf(t / res, 0.0f);
}

inline __device__ float advance_to_next_voxel(float t, float cone_angle, const Vector3f& pos, const Vector3f& dir, const Vector3f& idir, uint32_t res) {
	// Analytic stepping by a multiple of dt. Make empty space unequal to non-empty space
	// due to the different stepping.
	// float dt = calc_dt(t, cone_angle);
	// return t + ceilf(fmaxf(distance_to_next_voxel(pos, dir, idir, res) / dt, 0.5f)) * dt;

	// Regular stepping (may be slower but matches non-empty space)
	float t_target = t + distance_to_next_voxel(pos, dir, idir, res);
	do {
		t += calc_dt(t, cone_angle);
	} while (t < t_target);
	return t;
}

__device__ float network_to_rgb(float val, ENerfActivation activation) {
	switch (activation) {
		case ENerfActivation::None: return val;
		case ENerfActivation::ReLU: return val > 0.0f ? val : 0.0f;
		case ENerfActivation::Logistic: return tcnn::logistic(val);
		case ENerfActivation::Exponential: return __expf(tcnn::clamp(val, -10.0f, 10.0f));
		default: assert(false);
	}
	return 0.0f;
}

__device__ float network_to_rgb_derivative(float val, ENerfActivation activation) {
	switch (activation) {
		case ENerfActivation::None: return 1.0f;
		case ENerfActivation::ReLU: return val > 0.0f ? 1.0f : 0.0f;
		case ENerfActivation::Logistic: { float density = tcnn::logistic(val); return density * (1 - density); };
		case ENerfActivation::Exponential: return __expf(tcnn::clamp(val, -10.0f, 10.0f));
		default: assert(false);
	}
	return 0.0f;
}

__device__ float network_to_density(float val, ENerfActivation activation) {
	switch (activation) {
		case ENerfActivation::None: return val;
		case ENerfActivation::ReLU: return val > 0.0f ? val : 0.0f;
		case ENerfActivation::Logistic: return tcnn::logistic(val);
		case ENerfActivation::Exponential: return __expf(val);
		default: assert(false);
	}
	return 0.0f;
}

__device__ float network_to_density_derivative(float val, ENerfActivation activation) {
	switch (activation) {
		case ENerfActivation::None: return 1.0f;
		case ENerfActivation::ReLU: return val > 0.0f ? 1.0f : 0.0f;
		case ENerfActivation::Logistic: { float density = tcnn::logistic(val); return density * (1 - density); };
		case ENerfActivation::Exponential: return __expf(tcnn::clamp(val, -15.0f, 15.0f));
		default: assert(false);
	}
	return 0.0f;
}

__device__ Array3f network_to_rgb(const tcnn::vector_t<tcnn::network_precision_t, 4>& local_network_output, ENerfActivation activation) {
	return {
		network_to_rgb(float(local_network_output[0]), activation),
		network_to_rgb(float(local_network_output[1]), activation),
		network_to_rgb(float(local_network_output[2]), activation)
	};
}

__device__ Vector3f warp_position(const Vector3f& pos, const BoundingBox& aabb) {
	// return {tcnn::logistic(pos.x() - 0.5f), tcnn::logistic(pos.y() - 0.5f), tcnn::logistic(pos.z() - 0.5f)};
	// return pos;

	return aabb.relative_pos(pos);
}

__device__ Vector3f unwarp_position(const Vector3f& pos, const BoundingBox& aabb) {
	// return {logit(pos.x()) + 0.5f, logit(pos.y()) + 0.5f, logit(pos.z()) + 0.5f};
	// return pos;

	return aabb.min + pos.cwiseProduct(aabb.diag());
}

__device__ Vector3f unwarp_position_derivative(const Vector3f& pos, const BoundingBox& aabb) {
	// return {logit(pos.x()) + 0.5f, logit(pos.y()) + 0.5f, logit(pos.z()) + 0.5f};
	// return pos;

	return aabb.diag();
}

__device__ Vector3f warp_position_derivative(const Vector3f& pos, const BoundingBox& aabb) {
	return unwarp_position_derivative(pos, aabb).cwiseInverse();
}

__device__ Vector3f warp_direction(const Vector3f& dir) {
	return (dir + Vector3f::Ones()) * 0.5f;
}

__device__ Vector3f unwarp_direction(const Vector3f& dir) {
	return dir * 2.0f - Vector3f::Ones();
}

__device__ Vector3f warp_direction_derivative(const Vector3f& dir) {
	return Vector3f::Constant(0.5f);
}

__device__ Vector3f unwarp_direction_derivative(const Vector3f& dir) {
	return Vector3f::Constant(2.0f);
}

__device__ float warp_dt(float dt) {
	float max_stepsize = MIN_CONE_STEPSIZE() * (1<<(NERF_CASCADES()-1));
	return (dt - MIN_CONE_STEPSIZE()) / (max_stepsize - MIN_CONE_STEPSIZE());
}

__device__ float unwarp_dt(float dt) {
	float max_stepsize = MIN_CONE_STEPSIZE() * (1<<(NERF_CASCADES()-1));
	return dt * (max_stepsize - MIN_CONE_STEPSIZE()) + MIN_CONE_STEPSIZE();
}

__device__ uint32_t cascaded_grid_idx_at(Vector3f pos, uint32_t mip) {
	float mip_scale = scalbnf(1.0f, -mip);
	pos -= Vector3f::Constant(0.5f);
	pos *= mip_scale;
	pos += Vector3f::Constant(0.5f);

	Vector3i i = (pos * NERF_GRIDSIZE()).cast<int>();

	if (i.x() < -1 || i.x() > NERF_GRIDSIZE() || i.y() < -1 || i.y() > NERF_GRIDSIZE() || i.z() < -1 || i.z() > NERF_GRIDSIZE()) {
		printf("WTF %d %d %d\n", i.x(), i.y(), i.z());
	}

	uint32_t idx = tcnn::morton3D(
		tcnn::clamp(i.x(), 0, (int)NERF_GRIDSIZE()-1),
		tcnn::clamp(i.y(), 0, (int)NERF_GRIDSIZE()-1),
		tcnn::clamp(i.z(), 0, (int)NERF_GRIDSIZE()-1)
	);

	return idx;
}

__device__ bool density_grid_occupied_at(const Vector3f& pos, const uint8_t* density_grid_bitfield, uint32_t mip) {
	uint32_t idx = cascaded_grid_idx_at(pos, mip);
	return density_grid_bitfield[idx/8+grid_mip_offset(mip)/8] & (1<<(idx%8));
}

__device__ float cascaded_grid_at(Vector3f pos, const float* cascaded_grid, uint32_t mip) {
	uint32_t idx = cascaded_grid_idx_at(pos, mip);
	return cascaded_grid[idx+grid_mip_offset(mip)];
}

__device__ float& cascaded_grid_at(Vector3f pos, float* cascaded_grid, uint32_t mip) {
	uint32_t idx = cascaded_grid_idx_at(pos, mip);
	return cascaded_grid[idx+grid_mip_offset(mip)];
}

__global__ void extract_srgb_with_activation(const uint32_t n_elements,	const uint32_t rgb_stride, const float* __restrict__ rgbd, float* __restrict__ rgb, ENerfActivation rgb_activation, bool from_linear) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	const uint32_t elem_idx = i / 3;
	const uint32_t dim_idx = i - elem_idx * 3;

	float c = network_to_rgb(rgbd[elem_idx*4 + dim_idx], rgb_activation);
	if (from_linear) {
		c = linear_to_srgb(c);
	}
	rgb[elem_idx*rgb_stride + dim_idx] = c;
}

__global__ void mark_untrained_density_grid(const uint32_t n_elements,  float* __restrict__ grid_out,
	const uint32_t n_training_images,
	const Vector2f* __restrict__ focal_lengths,
	const Matrix<float, 3, 4>* training_xforms,
	Vector2i resolution
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;
	uint32_t level = i / (NERF_GRIDSIZE()*NERF_GRIDSIZE()*NERF_GRIDSIZE());
	uint32_t pos_idx = i % (NERF_GRIDSIZE()*NERF_GRIDSIZE()*NERF_GRIDSIZE());

	uint32_t x = tcnn::morton3D_invert(pos_idx>>0);
	uint32_t y = tcnn::morton3D_invert(pos_idx>>1);
	uint32_t z = tcnn::morton3D_invert(pos_idx>>2);

	float half_resx=resolution.x()*0.5f;
	float half_resy=resolution.y()*0.5f;

	Vector3f pos = ((Vector3f{(float)x+0.5f, (float)y+0.5f, (float)z+0.5f}) / NERF_GRIDSIZE() - Vector3f::Constant(0.5f)) * scalbnf(1.0f, level) + Vector3f::Constant(0.5f);
	float voxel_radius = 0.5f*SQRT3()*scalbnf(1.0f, level) / NERF_GRIDSIZE();
	int count=0;
	for (uint32_t j=0; j < n_training_images; ++j) {
		Matrix<float, 3, 4> xform = training_xforms[j];
		Vector3f ploc = pos-xform.col(3);
		float x=ploc.dot(xform.col(0));
		float y=ploc.dot(xform.col(1));
		float z=ploc.dot(xform.col(2));
		if (z>0.f) {
			auto focal = focal_lengths[j];
			// TODO - add a box / plane intersection to stop thomas from murdering me
			if (fabsf(x)-voxel_radius < z/focal.x()*half_resx && fabsf(y)-voxel_radius < z/focal.y()*half_resy) {
				count++;
				if (count > 0) break;
			}
		}
	}
	grid_out[i] = (count > 0) ? 0.f : -1.f;
}


__global__ void generate_grid_samples_nerf_uniform(Eigen::Vector3i res_3d, const uint32_t step, BoundingBox render_aabb, BoundingBox train_aabb, NerfPosition* __restrict__ out) {
	// check grid_in for negative values -> must be negative on output
	uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	uint32_t z = threadIdx.z + blockIdx.z * blockDim.z;
	if (x>=res_3d.x() || y>=res_3d.y() || z>=res_3d.z())
		return;
	uint32_t i = x+ y*res_3d.x() + z*res_3d.x()*res_3d.y();
	Vector3f pos = Array3f{(float)x, (float)y, (float)z} * Array3f{1.f/res_3d.x(),1.f/res_3d.y(),1.f/res_3d.z()};
	pos = pos.cwiseProduct(render_aabb.max - render_aabb.min) + render_aabb.min;
	out[i] = { warp_position(pos, train_aabb), warp_dt(MIN_CONE_STEPSIZE()) };
}

// generate samples for uniform grid including constant ray direction
__global__ void generate_grid_samples_nerf_uniform_dir(Eigen::Vector3i res_3d, const uint32_t step, BoundingBox render_aabb, BoundingBox train_aabb, Eigen::Vector3f ray_dir, NerfCoordinate* __restrict__ network_input) {
	// check grid_in for negative values -> must be negative on output
	uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	uint32_t z = threadIdx.z + blockIdx.z * blockDim.z;
	if (x>=res_3d.x() || y>=res_3d.y() || z>=res_3d.z())
		return;
	uint32_t i = x+ y*res_3d.x() + z*res_3d.x()*res_3d.y();
	Vector3f pos = Array3f{(float)x, (float)y, (float)z} * Array3f{1.f/res_3d.x(),1.f/res_3d.y(),1.f/res_3d.z()};
	pos = pos.cwiseProduct(render_aabb.max - render_aabb.min) + render_aabb.min;
	network_input[i] = { warp_position(pos, train_aabb), warp_direction(ray_dir), warp_dt(MIN_CONE_STEPSIZE()) };
}

inline __device__ int mip_from_pos(const Vector3f& pos) {
	int exponent;
	float maxval = (pos - Vector3f::Constant(0.5f)).cwiseAbs().maxCoeff();
	frexpf(maxval, &exponent);
	return min(NERF_CASCADES()-1, max(0, exponent+1));
}

inline __device__ int mip_from_dt(float dt, const Vector3f& pos) {
	int mip = mip_from_pos(pos);
	dt *= 2*NERF_GRIDSIZE();
	if (dt<1.f) return mip;
	int exponent;
	frexpf(dt, &exponent);
	return min(NERF_CASCADES()-1, max(exponent, mip));
}


__global__ void generate_grid_samples_nerf_nonuniform(const uint32_t n_elements, default_rng_t rng, const uint32_t step, BoundingBox aabb, const float* __restrict__ grid_in, NerfPosition* __restrict__ out, uint32_t* __restrict__ indices, uint32_t n_cascades, float thresh) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	// 1 random number to select the level, 3 to select the position.
	rng.advance(i*4);
	uint32_t level = (uint32_t)(random_val(rng) * n_cascades) % n_cascades;

	// Select grid cell that has density
	uint32_t idx;
	for (uint32_t j = 0; j < 10; ++j) {
		idx = ((i+step*n_elements) * 56924617 + j * 19349663 + 96925573) % (NERF_GRIDSIZE()*NERF_GRIDSIZE()*NERF_GRIDSIZE());
		idx += level * NERF_GRIDSIZE()*NERF_GRIDSIZE()*NERF_GRIDSIZE();
		if (grid_in[idx] > thresh) {
			break;
		}
	}

	// Random position within that cellq
	uint32_t pos_idx = idx % (NERF_GRIDSIZE()*NERF_GRIDSIZE()*NERF_GRIDSIZE());

	uint32_t x = tcnn::morton3D_invert(pos_idx>>0);
	uint32_t y = tcnn::morton3D_invert(pos_idx>>1);
	uint32_t z = tcnn::morton3D_invert(pos_idx>>2);

	Vector3f pos = ((Vector3f{(float)x, (float)y, (float)z} + random_val_3d(rng)) / NERF_GRIDSIZE() - Vector3f::Constant(0.5f)) * scalbnf(1.0f, level) + Vector3f::Constant(0.5f);

	out[i] = { warp_position(pos, aabb), warp_dt(MIN_CONE_STEPSIZE()) };
	indices[i] = idx;
}

__global__ void splat_grid_samples_nerf_max_nearest_neighbor(const uint32_t n_elements, const uint32_t* __restrict__ indices, int padded_output_width, const tcnn::network_precision_t* network_output, float* __restrict__ grid_out, ENerfActivation rgb_activation, ENerfActivation density_activation) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	uint32_t local_idx = indices[i];

	// Current setting: optical thickness of the smallest possible stepsize.
	// Uncomment for:   optical thickness of the ~expected step size when the observer is in the middle of the scene
	uint32_t level = 0;//local_idx / (NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_GRIDSIZE());

	float mlp = network_to_density(float(network_output[i * padded_output_width]), density_activation);
	float optical_thickness = mlp * scalbnf(MIN_CONE_STEPSIZE(), level);

	// Positive floats are monotonically ordered when their bit pattern is interpretes as uint.
	// uint atomicMax is thus perfectly acceptable.
	atomicMax((uint32_t*)&grid_out[local_idx], __float_as_uint(optical_thickness));
}

__global__ void grid_samples_half_to_float(const uint32_t n_elements, BoundingBox aabb, float *dst, int padded_output_width, const tcnn::network_precision_t* network_output, ENerfActivation density_activation, const NerfPosition* __restrict__ coords_in, const float* __restrict__ grid_in) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	// let's interpolate for marching cubes based on the raw MLP output, not the density (exponentiated) version
	//float mlp = network_to_density(float(network_output[i * padded_output_width]), density_activation);
	float mlp = float(network_output[i * padded_output_width]);

	if (grid_in) {
		Vector3f pos = unwarp_position(coords_in[i].p, aabb);
		float grid_density = cascaded_grid_at(pos, grid_in, mip_from_pos(pos));
		if (grid_density < NERF_MIN_OPTICAL_THICKNESS()) {
			mlp = -10000.f;
		}
	}
	dst[i] = mlp;
}

__global__ void ema_grid_samples_nerf(const uint32_t n_elements,
	float decay,
	const uint32_t count,
	float* __restrict__ grid_out,
	const float* __restrict__ grid_in
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	float importance = grid_in[i];

	// float ema_debias_old = 1 - (float)powf(decay, count);
	// float ema_debias_new = 1 - (float)powf(decay, count+1);

	// float filtered_val = ((grid_out[i] * decay * ema_debias_old + importance * (1 - decay)) / ema_debias_new);
	// grid_out[i] = filtered_val;

	// Maximum instead of EMA allows capture of very thin features.
	// Basically, we want the grid cell turned on as soon as _ANYTHING_ visible is in there.

	float prev_val = grid_out[i];
	float val = (prev_val<0.f) ? prev_val : fmaxf(prev_val * decay, importance);
	grid_out[i] = val;
}

__global__ void decay_sharpness_grid_nerf(const uint32_t n_elements, float decay, float* __restrict__ grid) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;
	grid[i] *= decay;
}

__global__ void grid_to_bitfield(const uint32_t n_elements,
	const float* __restrict__ grid,
	uint8_t* __restrict__ grid_bitfield,
	const float* __restrict__ mean_density_ptr
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	uint8_t bits = 0;

	float thresh = std::min(NERF_MIN_OPTICAL_THICKNESS(), *mean_density_ptr);

	#pragma unroll
	for (uint8_t j = 0; j < 8; ++j) {
		bits |= grid[i*8+j] > thresh ? ((uint8_t)1 << j) : 0;
	}

	grid_bitfield[i] = bits;
}

__global__ void bitfield_max_pool(const uint32_t n_elements,
	const uint8_t* __restrict__ prev_level,
	uint8_t* __restrict__ next_level
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	uint8_t bits = 0;

	#pragma unroll
	for (uint8_t j = 0; j < 8; ++j) {
		// If any bit is set in the previous level, set this
		// level's bit. (Max pooling.)
		bits |= prev_level[i*8+j] > 0 ? ((uint8_t)1 << j) : 0;
	}

	uint32_t x = tcnn::morton3D_invert(i>>0) + NERF_GRIDSIZE()/8;
	uint32_t y = tcnn::morton3D_invert(i>>1) + NERF_GRIDSIZE()/8;
	uint32_t z = tcnn::morton3D_invert(i>>2) + NERF_GRIDSIZE()/8;

	next_level[tcnn::morton3D(x, y, z)] |= bits;
}


__global__ void advance_pos_nerf(
	const uint32_t n_elements,
	BoundingBox render_aabb,
	Vector3f camera_fwd,
	Vector2f focal_length,
	uint32_t spp,
	NerfPayload* __restrict__ payloads,
	const uint8_t* __restrict__ density_grid,
	uint32_t min_mip,
	float cone_angle_constant
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	NerfPayload& payload = payloads[i];

	if (!payload.alive) {
		return;
	}

	Vector3f origin = payload.origin;
	Vector3f dir = payload.dir;
	Vector3f idir = dir.cwiseInverse();

	float cone_angle = calc_cone_angle(dir.dot(camera_fwd), focal_length, cone_angle_constant);

	float t = payload.t;
	float dt = calc_dt(t, cone_angle);
	t += ld_random_val(spp, i * 786433) * dt;
	Vector3f pos;

	while (1) {
		if (!render_aabb.contains(pos = origin + dir * t)) {
			payload.alive = false;
			break;
		}

		dt = calc_dt(t, cone_angle);
		uint32_t mip = max(min_mip, mip_from_dt(dt, pos));

		if (!density_grid || density_grid_occupied_at(pos, density_grid, mip)) {
			break;
		}

		uint32_t res = NERF_GRIDSIZE()>>mip;
		t = advance_to_next_voxel(t, cone_angle, pos, dir, idir, res);
	}

	payload.t = t;
}

__global__ void generate_nerf_network_inputs_from_positions(const uint32_t n_elements, BoundingBox aabb, const Vector3f* __restrict__ pos, NerfCoordinate* __restrict__ network_input) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	Vector3f dir=(pos[i]-Vector3f::Constant(0.5f)).normalized(); // choose outward pointing directions, for want of a better choice
	network_input[i] = { warp_position(pos[i], aabb), warp_direction(dir), warp_dt(MIN_CONE_STEPSIZE()) };
}

__global__ void generate_nerf_network_inputs_at_current_position(const uint32_t n_elements, BoundingBox aabb, const NerfPayload* __restrict__ payloads, NerfCoordinate* __restrict__ network_input) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	Vector3f dir = payloads[i].dir;
	network_input[i] = { warp_position(payloads[i].origin + dir * payloads[i].t, aabb), warp_direction(dir), warp_dt(MIN_CONE_STEPSIZE()) };
}

__global__ void compute_nerf_density(const uint32_t n_elements, Array4f* network_output, ENerfActivation rgb_activation, ENerfActivation density_activation) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	Array4f rgba = network_output[i];
	rgba.w() = tcnn::clamp(1.f - __expf(-network_to_density(rgba.w(), density_activation) / 100.0f), 0.0f, 1.0f);
	rgba.x() = network_to_rgb(rgba.x(), rgb_activation) * rgba.w();
	rgba.y() = network_to_rgb(rgba.y(), rgb_activation) * rgba.w();
	rgba.z() = network_to_rgb(rgba.z(), rgb_activation) * rgba.w();

	network_output[i] = rgba;
}

__global__ void generate_next_nerf_network_inputs(
	const uint32_t n_elements,
	BoundingBox render_aabb,
	BoundingBox train_aabb,
	Vector2f focal_length,
	Vector3f camera_fwd,
	NerfPayload* __restrict__ payloads,
	NerfCoordinate* __restrict__ network_input,
	uint32_t n_steps,
	const uint8_t* __restrict__ density_grid,
	uint32_t min_mip,
	float cone_angle_constant
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	NerfPayload& payload = payloads[i];

	if (!payload.alive) {
		return;
	}

	Vector3f origin = payload.origin;
	Vector3f dir = payload.dir;
	Vector3f idir = dir.cwiseInverse();

	float cone_angle = calc_cone_angle(dir.dot(camera_fwd), focal_length, cone_angle_constant);

	float t = payload.t;

	for (uint32_t j = 0; j < n_steps; ++j) {
		Vector3f pos;
		float dt = 0.0f;
		while (1) {
			if (!render_aabb.contains(pos = origin + dir * t)) {
				payload.n_steps = j;
				return;
			}

			dt = calc_dt(t, cone_angle);
			uint32_t mip = max(min_mip, mip_from_dt(dt, pos));

			if (!density_grid || density_grid_occupied_at(pos, density_grid, mip)) {
				break;
			}

			uint32_t res = NERF_GRIDSIZE()>>mip;
			t = advance_to_next_voxel(t, cone_angle, pos, dir, idir, res);
		}

		network_input[i*n_steps + j] = { warp_position(pos, train_aabb), warp_direction(dir), warp_dt(dt) }; // XXXCONE
		t += dt;
	}

	payload.t = t;
	payload.n_steps = n_steps;
}


__global__ void composite_kernel_nerf(
	const uint32_t n_elements,
	const uint32_t current_step,
	BoundingBox aabb,
	const uint32_t n_training_images,
	const Matrix<float, 3, 4>* training_xforms,
	Matrix<float, 3, 4> camera_matrix,
	Vector2f focal_length,
	float depth_scale,
	Array4f* rgba,
	NerfPayload* payloads,
	const NerfCoordinate* network_input,
	const tcnn::network_precision_t* network_output,
	uint32_t padded_output_width,
	uint32_t n_steps,
	ERenderMode render_mode,
	const uint8_t* density_grid,
	ENerfActivation rgb_activation,
	ENerfActivation density_activation,
	int show_accel,
	float min_alpha
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	NerfPayload& payload = payloads[i];

	if (!payload.alive) {
		return;
	}

	Array4f local_rgba = rgba[i];
	Vector3f origin = payload.origin;
	Vector3f cam_fwd = camera_matrix.col(2);
	// Composite in the last n steps
	uint32_t actual_n_steps = payload.n_steps;
	uint32_t j = 0;

	for (; j < actual_n_steps; ++j) {
		const tcnn::vector_t<tcnn::network_precision_t, 4> local_network_output = *(tcnn::vector_t<tcnn::network_precision_t, 4>*)&network_output[(i*n_steps+j)*padded_output_width];
		Vector3f warped_pos = network_input[i*n_steps+j].pos.p;
		Vector3f pos = unwarp_position(warped_pos, aabb);

		//Vector3f pos2 = pos-Vector3f::Constant(0.5f);
		//float fog_scale = 1.f/max(1.f,pos2.dot(pos2)*4.f);

		float T = 1.f - local_rgba.w();
		float dt = unwarp_dt(network_input[i*n_steps+j].dt);
		float alpha = 1.f - __expf(-network_to_density(float(local_network_output[3]), density_activation) * dt /* * fog_scale*/);
		if (show_accel>=0)
		alpha=1.f;
		float weight = alpha * T;

		Array3f rgb = network_to_rgb(local_network_output, rgb_activation);

		if (render_mode == ERenderMode::Normals) {
			// Network input contains the gradient of the network output w.r.t. input.
			// So to compute density gradients, we need to apply the chain rule.
			// The normal is then in the opposite direction of the density gradient (i.e. the direction of decreasing density)
			Vector3f normal = -network_to_density_derivative(float(local_network_output[3]), density_activation) * warped_pos;
			rgb = normal.normalized().array();
		} else if (render_mode == ERenderMode::Positions || render_mode == ERenderMode::EncodingVis) {
			if (show_accel>=0) {
				uint32_t mip = max(show_accel, mip_from_pos(pos));
				uint32_t res = NERF_GRIDSIZE() >> mip;
				int ix = pos.x()*(res);
				int iy = pos.y()*(res);
				int iz = pos.z()*(res);
				default_rng_t rng(ix+iy*232323+iz*727272);
				rgb.x() = 1.f-mip*(1.f/(NERF_CASCADES()-1));
				rgb.y() = rng.next_float();
				rgb.z() = rng.next_float();
			} else {
				rgb = pos.array();
			}
		} else if (render_mode == ERenderMode::Depth) {
			float z=cam_fwd.dot(pos-origin) * depth_scale;
			rgb = {z,z,z};
		} else if (render_mode == ERenderMode::Distance) {
			float z=(pos-origin).norm() * depth_scale;
			rgb = {z,z,z};
		} else if (render_mode == ERenderMode::Stepsize) {
			float warped_dt = warp_dt(dt);
			rgb = {warped_dt,warped_dt,warped_dt};
		} else if (render_mode == ERenderMode::AO) {
			rgb = Array3f::Constant(alpha);
		}

		local_rgba.head<3>() += rgb * weight;
		local_rgba.w() += weight;

		if (local_rgba.w() > (1.0f - min_alpha)) {
			rgba[i] = local_rgba / local_rgba.w();
			break;
		}
	}

	if (j < n_steps) {
		payload.alive = false;
		payload.n_steps = j + current_step;
	}

	rgba[i] = local_rgba;
}

static constexpr float UNIFORM_SAMPLING_FRACTION = 0.5f;

inline __device__ Vector2f sample_cdf_2d(Vector2f sample, uint32_t img, const Vector2i& res, const float* __restrict__ cdf_x_cond_y, const float* __restrict__ cdf_y, float* __restrict__ pdf) {
	if (sample.x() < UNIFORM_SAMPLING_FRACTION) {
		sample.x() /= UNIFORM_SAMPLING_FRACTION;
		return sample;
	}

	sample.x() = (sample.x() - UNIFORM_SAMPLING_FRACTION) / (1.0f - UNIFORM_SAMPLING_FRACTION);

	cdf_y += img * res.y();

	// First select row according to cdf_y
	uint32_t y = binary_search(sample.y(), cdf_y, res.y());
	float prev = y > 0 ? cdf_y[y-1] : 0.0f;
	float pmf_y = cdf_y[y] - prev;
	sample.y() = (sample.y() - prev) / pmf_y;

	cdf_x_cond_y += img * res.y() * res.x() + y * res.x();

	// Then, select col according to x
	uint32_t x = binary_search(sample.x(), cdf_x_cond_y, res.x());
	prev = x > 0 ? cdf_x_cond_y[x-1] : 0.0f;
	float pmf_x = cdf_x_cond_y[x] - prev;
	sample.x() = (sample.x() - prev) / pmf_x;

	if (pdf) {
		*pdf = pmf_x * pmf_y * res.prod();
	}

	return {((float)x + sample.x()) / (float)res.x(), ((float)y + sample.y()) / (float)res.y()};
}

inline __device__ float pdf_2d(Vector2f sample, uint32_t img, const Vector2i& res, const float* __restrict__ cdf_x_cond_y, const float* __restrict__ cdf_y) {
	Vector2i p = (sample.cwiseProduct(res.cast<float>())).cast<int>().cwiseMax(0).cwiseMin(res - Vector2i::Ones());

	cdf_y += img * res.y();
	cdf_x_cond_y += img * res.y() * res.x() + p.y() * res.x();

	float pmf_y = cdf_y[p.y()];
	if (p.y() > 0) {
		pmf_y -= cdf_y[p.y()-1];
	}

	float pmf_x = cdf_x_cond_y[p.x()];
	if (p.x() > 0) {
		pmf_x -= cdf_x_cond_y[p.x()-1];
	}

	// Probability mass of picking the pixel
	float pmf = pmf_x * pmf_y;

	// To convert to probability density, divide by area of pixel
	return UNIFORM_SAMPLING_FRACTION + pmf * res.prod() * (1.0f - UNIFORM_SAMPLING_FRACTION);
}

inline __device__ Vector2f nerf_random_image_pos_training(default_rng_t& rng, const Vector2i& resolution, bool snap_to_pixel_centers, const float* __restrict__ cdf_x_cond_y, const float* __restrict__ cdf_y, const Vector2i& cdf_res, uint32_t img, float* __restrict__ pdf = nullptr) {
	Vector2f xy = random_val_2d(rng);

	if (cdf_x_cond_y) {
		xy = sample_cdf_2d(xy, img, cdf_res, cdf_x_cond_y, cdf_y, pdf);
	} else if (pdf) {
		*pdf = 1.0f;
	}

	if (snap_to_pixel_centers) {
		xy = (xy.cwiseProduct(resolution.cast<float>()).cast<int>().cwiseMax(0).cwiseMin(resolution - Vector2i::Ones()).cast<float>() + Vector2f::Constant(0.5f)).cwiseQuotient(resolution.cast<float>());
	}
	return xy;
}

inline __device__ uint32_t image_idx(uint32_t base_idx, uint32_t n_rays, uint32_t n_rays_total, uint32_t n_training_images, const float* __restrict__ cdf = nullptr, float* __restrict__ pdf = nullptr) {
	if (cdf) {
		float sample = ld_random_val(base_idx + n_rays_total, 0xdeadbeef);
		// float sample = random_val(base_idx + n_rays_total);
		uint32_t img = binary_search(sample, cdf, n_training_images);

		if (pdf) {
			float prev = img > 0 ? cdf[img-1] : 0.0f;
			*pdf = (cdf[img] - prev) * n_training_images;
		}

		return img;
	}

	// return ((base_idx + n_rays_total) * 56924617 + 96925573) % n_training_images;

	// Neighboring threads in the warp process the same image. Increases locality.
	if (pdf) {
		*pdf = 1.0f;
	}
	return (((base_idx + n_rays_total) * n_training_images) / n_rays) % n_training_images;
}

inline __device__ Vector2i image_pos(const Vector2f& pos, const Vector2i& resolution) {
	return pos.cwiseProduct(resolution.cast<float>()).cast<int>().cwiseMin(resolution - Vector2i::Constant(1)).cwiseMax(0);
}

inline __device__ uint64_t pixel_idx(const Vector2i& pos, const Vector2i& resolution, uint32_t img) {
	return pos.x() + pos.y() * resolution.x() + img * (uint64_t)resolution.x() * resolution.y();
}

inline __device__ uint64_t pixel_idx(const Vector2f& xy, const Vector2i& resolution, uint32_t img) {
	return pixel_idx(image_pos(xy, resolution), resolution, img);
}

__global__ void generate_training_samples_nerf(
	const uint32_t n_rays,
	BoundingBox aabb,
	const uint32_t max_samples,
	const uint32_t n_rays_total,
	default_rng_t rng,
	const Ray* __restrict__ rays_in,
	uint32_t* __restrict__ ray_counter,
	uint32_t* __restrict__ numsteps_counter,
	uint32_t* __restrict__ ray_indices_out,
	Ray* __restrict__ rays_out,
	uint32_t* __restrict__ numsteps_out,
	NerfCoordinate* __restrict__ coords_out,
	Vector2i resolution,
	const uint32_t n_training_images,
	Vector2f principal_point,
	const Vector2f* __restrict__ focal_lengths,
	const Matrix<float, 3, 4>* training_xforms,
	CameraDistortion camera_distortion,
	const uint8_t* __restrict__ density_grid,
	bool max_level_rand_training,
	float* __restrict__ max_level_ptr,
	bool snap_to_pixel_centers,
	bool train_envmap,
	float cone_angle_constant,
	const float* __restrict__ distortion_data,
	const Vector2i distortion_resolution,
	const float* __restrict__ cdf_x_cond_y,
	const float* __restrict__ cdf_y,
	const float* __restrict__ cdf_img,
	const Vector2i cdf_res,
	float near_distance,
	const __half* __restrict__ training_images
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_rays) return;

	uint32_t img = image_idx(i, n_rays, n_rays_total, n_training_images, cdf_img);

	rng.advance(i * N_MAX_RANDOM_SAMPLES_PER_RAY());
	Vector2f xy = nerf_random_image_pos_training(rng, resolution, snap_to_pixel_centers, cdf_x_cond_y, cdf_y, cdf_res, img);

	// Negative values indicate masked-away regions
	if ((float)training_images[pixel_idx(xy, resolution, img)*4] < 0.0f) {
		return;
	}

	float max_level = max_level_rand_training ? (random_val(rng) * 2.0f) : 1.0f; // Multiply by 2 to ensure 50% of training is at max level

	Matrix<float, 3, 4> xform = training_xforms[img];
	Vector2f focal_length = focal_lengths[img];

	Ray ray;

	if (rays_in) {
		// Rays have been explicitly supplied. Read them.
		ray = rays_in[pixel_idx(xy, resolution, img)];
	} else {
		// Rays need to be inferred from the camera matrix
		ray.o = xform.col(3);
		ray.d = {
			(xy.x()-principal_point.x())*resolution.x() / focal_length.x(),
			(xy.y()-principal_point.y())*resolution.y() / focal_length.y(),
			1.0f,
		};
		if (!camera_distortion.is_zero()) {
			iterative_camera_undistortion((float*)&camera_distortion, &ray.d.x(), &ray.d.y());
		}
		if (distortion_data) {
			ray.d.head<2>() += read_image<2>(distortion_data, distortion_resolution, xy);
		}

		ray.d = (xform.block<3, 3>(0, 0) * ray.d).normalized();
	}

	Vector2f tminmax = aabb.ray_intersect(ray.o, ray.d);
	float cone_angle = calc_cone_angle(ray.d.dot(xform.col(2)), focal_length, cone_angle_constant);

	// The near distance prevents learning of camera-specific fudge right in front of the camera
	tminmax.x() = fmaxf(tminmax.x(), near_distance);

	float startt = tminmax.x();
	startt += calc_dt(startt, cone_angle) * random_val(rng);
	Vector3f idir = ray.d.cwiseInverse();

	// first pass to compute an accurate number of steps
	uint32_t j = 0;
	float t=startt;
	Vector3f pos;

	while (aabb.contains(pos = ray.o + t * ray.d) && j < NERF_STEPS()) {
		float dt = calc_dt(t, cone_angle);
		uint32_t mip = mip_from_dt(dt, pos);
		if (density_grid_occupied_at(pos, density_grid, mip)) {
			++j;
			t += dt;
		} else {
			uint32_t res = NERF_GRIDSIZE()>>mip;
			t = advance_to_next_voxel(t, cone_angle, pos, ray.d, idir, res);
		}
	}
	if (j == 0 && !train_envmap) {
		return;
	}
	uint32_t numsteps = j;
	uint32_t base = atomicAdd(numsteps_counter, numsteps);	 // first entry in the array is a counter
	if (base + numsteps > max_samples) {
		return;
	}

	coords_out += base;

	uint32_t ray_idx = atomicAdd(ray_counter, 1);

	ray_indices_out[ray_idx] = i;
	rays_out[ray_idx] = ray;
	numsteps_out[ray_idx*2+0] = numsteps;
	numsteps_out[ray_idx*2+1] = base;

	Vector3f warped_dir = warp_direction(ray.d);
	t=startt;
	j=0;
	while (aabb.contains(pos = ray.o + t * ray.d) && j < numsteps) {
		float dt = calc_dt(t, cone_angle);
		uint32_t mip = mip_from_dt(dt, pos);
		if (density_grid_occupied_at(pos, density_grid, mip)) {
			coords_out[j] = { warp_position(pos, aabb), warped_dir, warp_dt(dt) };
			++j;
			t += dt;
		} else {
			uint32_t res = NERF_GRIDSIZE()>>mip;
			t = advance_to_next_voxel(t, cone_angle, pos, ray.d, idir, res);
		}
	}
	if (max_level_rand_training) {
		max_level_ptr += base;
		for (j = 0; j < numsteps; ++j) {
			max_level_ptr[j] = max_level;
		}
	}
}

__device__ LossAndGradient loss_and_gradient(const Vector3f& target, const Vector3f& prediction, ELossType loss_type) {
	switch (loss_type) {
		case ELossType::RelativeL2:  return relative_l2_loss(target, prediction); break;
		case ELossType::L1:          return l1_loss(target, prediction); break;
		case ELossType::Mape:        return mape_loss(target, prediction); break;
		case ELossType::Smape:       return smape_loss(target, prediction); break;
		// Note: we divide the huber loss by a factor of 5 such that its L2 region near zero
		// matches with the L2 loss and error numbers become more comparable. This allows reading
		// off dB numbers of ~converged models and treating them as approximate PSNR to compare
		// with other NeRF methods. Self-normalizing optimizers such as Adam are agnostic to such
		// constant factors; optimization is therefore unaffected.
		case ELossType::Huber:       return huber_loss(target, prediction, 0.1f) / 5.0f; break;
		case ELossType::LogL1:       return log_l1_loss(target, prediction); break;
		default: case ELossType::L2: return l2_loss(target, prediction); break;
	}
}

inline __device__ Array3f composit_and_lerp(Vector2f pos, const Vector2i& resolution, uint32_t img, const __half* training_images, const Array3f& background_color, const Array3f& exposure_scale = Array3f::Ones()) {
	pos = (pos.cwiseProduct(resolution.cast<float>()) - Vector2f::Constant(0.5f)).cwiseMax(0.0f).cwiseMin(resolution.cast<float>() - Vector2f::Constant(1.0f + 1e-4f));

	Vector2i pos_int = pos.cast<int>();
	auto weight = pos - pos_int.cast<float>();

	Vector2i idx = pos_int.cwiseMin(resolution - Vector2i::Constant(2)).cwiseMax(0);

	auto read_val = [&](const Vector2i& p) {
		__half val[4];
		*(uint64_t*)&val[0] = ((uint64_t*)training_images)[pixel_idx(p, resolution, img)];
		return Array3f{val[0], val[1], val[2]} * exposure_scale + background_color * (1.0f - (float)val[3]);
	};

	Array3f result = (
		(1 - weight.x()) * (1 - weight.y()) * read_val({idx.x(), idx.y()}) +
		(weight.x()) * (1 - weight.y()) * read_val({idx.x()+1, idx.y()}) +
		(1 - weight.x()) * (weight.y()) * read_val({idx.x(), idx.y()+1}) +
		(weight.x()) * (weight.y()) * read_val({idx.x()+1, idx.y()+1})
	);

	return result;
}

inline __device__ Array3f composit(Vector2f pos, const Vector2i& resolution, uint32_t img, const __half* training_images, const Array3f& background_color, const Array3f& exposure_scale = Array3f::Ones()) {
	Vector2i idx = image_pos(pos, resolution);

	auto read_val = [&](const Vector2i& p) {
		__half val[4];
		*(uint64_t*)&val[0] = ((uint64_t*)training_images)[pixel_idx(p, resolution, img)];
		return Array3f{val[0], val[1], val[2]} * exposure_scale + background_color * (1.0f - (float)val[3]);
	};

	return read_val(idx);
}

inline __device__ Array4f read_rgba(Vector2f pos, const Vector2i& resolution, uint32_t img, const __half* training_images) {
	Vector2i idx = image_pos(pos, resolution);

	auto read_val = [&](const Vector2i& p) {
		__half val[4];
		*(uint64_t*)&val[0] = ((uint64_t*)training_images)[pixel_idx(p, resolution, img)];
		return Array4f{val[0], val[1], val[2], val[3]};
	};

	return read_val(idx);
}

__global__ void compute_loss_kernel_train_nerf(
	const uint32_t n_rays,
	BoundingBox aabb,
	const uint32_t n_rays_total,
	default_rng_t rng,
	const uint32_t max_samples_compacted,
	const uint32_t* __restrict__ rays_counter,
	float loss_scale,
	int padded_output_width,
	const float* __restrict__ envmap_data,
	float* __restrict__ envmap_gradient,
	const Vector2i envmap_resolution,
	ELossType envmap_loss_type,
	Array3f background_color,
	EColorSpace color_space,
	bool train_with_random_bg_color,
	bool train_in_linear_colors,
	const __half* __restrict__ training_images,
	const uint32_t n_training_images,
	Vector2i resolution,
	const tcnn::network_precision_t* network_output,
	uint32_t* __restrict__ numsteps_counter,
	const uint32_t* __restrict__ ray_indices_in,
	const Ray* __restrict__ rays_in,
	uint32_t* __restrict__ numsteps_in,
	const NerfCoordinate* __restrict__ coords_in,
	NerfCoordinate* __restrict__ coords_out,
	tcnn::network_precision_t* dloss_doutput,
	ELossType loss_type,
	float* __restrict__ loss_output,
	bool max_level_rand_training,
	float* __restrict__ max_level_compacted_ptr,
	ENerfActivation rgb_activation,
	ENerfActivation density_activation,
	bool snap_to_pixel_centers,
	float* __restrict__ error_map,
	const float* __restrict__ cdf_x_cond_y,
	const float* __restrict__ cdf_y,
	const float* __restrict__ cdf_img,
	const Vector2i error_map_res,
	const Vector2i error_map_cdf_res,
	const float* __restrict__ sharpness_data,
	Eigen::Vector2i sharpness_resolution,
	float* __restrict__ sharpness_grid,
	float* __restrict__ density_grid,
	const float* __restrict__ mean_density_ptr,
	const Eigen::Array3f* __restrict__ exposure,
	Eigen::Array3f* __restrict__ exposure_gradient
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= *rays_counter) { return; }

	// grab the number of samples for this ray, and the first sample
	uint32_t numsteps = numsteps_in[i*2+0];
	uint32_t base = numsteps_in[i*2+1];

	coords_in += base;
	network_output += base * 4;

	float T = 1.f;

	float EPSILON = 1e-4f;

	Array3f rgb_ray = Array3f::Zero();
	Vector3f hitpoint = Vector3f::Zero();

	uint32_t compacted_numsteps = 0;
	for (; compacted_numsteps < numsteps; ++compacted_numsteps) {
		if (T < EPSILON) {
			break;
		}

		const tcnn::vector_t<tcnn::network_precision_t, 4> local_network_output = *(tcnn::vector_t<tcnn::network_precision_t, 4>*)network_output;
		const Array3f rgb = network_to_rgb(local_network_output, rgb_activation);
		const Vector3f pos = unwarp_position(coords_in->pos.p, aabb);
		const float dt = unwarp_dt(coords_in->dt);

		float density = network_to_density(float(local_network_output[3]), density_activation);


		const float alpha = 1.f - __expf(-density * dt);
		const float weight = alpha * T;
		rgb_ray += weight * rgb;
		hitpoint += weight * pos;
		T *= (1.f - alpha);

		network_output += 4;
		coords_in++;
	}
	hitpoint /= (1.0f - T);

	// Must be same seed as above to obtain the same
	// background color.
	uint32_t ray_idx = ray_indices_in[i];
	rng.advance(ray_idx * N_MAX_RANDOM_SAMPLES_PER_RAY());

	float img_pdf = 1.0f;
	uint32_t img = image_idx(ray_idx, n_rays, n_rays_total, n_training_images, cdf_img, &img_pdf);

	float xy_pdf = 1.0f;
	Vector2f xy = nerf_random_image_pos_training(rng, resolution, snap_to_pixel_centers, cdf_x_cond_y, cdf_y, error_map_cdf_res, img, &xy_pdf);
	float max_level = max_level_rand_training ? (random_val(rng) * 2.0f) : 1.0f; // Multiply by 2 to ensure 50% of training is at max level

	if (train_with_random_bg_color) {
		background_color = random_val_3d(rng);
	}
	Array3f pre_envmap_background_color = background_color = srgb_to_linear(background_color);

	// Composit background behind envmap
	Array4f envmap_value;
	Vector3f dir;
	if (envmap_data) {
		dir = rays_in[i].d;
		envmap_value = read_envmap(envmap_data, envmap_resolution, dir);
		background_color = envmap_value.head<3>() + background_color * (1.0f - envmap_value.w());
	}

	Array3f exposure_scale = (0.6931471805599453f * exposure[img]).exp();
	// Array3f rgbtarget = composit_and_lerp(xy, resolution, img, training_images, background_color, exposure_scale);
	// Array3f rgbtarget = composit(xy, resolution, img, training_images, background_color, exposure_scale);
	Array4f texsamp = read_rgba(xy, resolution, img, training_images);

	Array3f rgbtarget;
	if (train_in_linear_colors || color_space == EColorSpace::Linear) {
		rgbtarget = exposure_scale * texsamp.head<3>() + (1.0f - texsamp.w()) * background_color;

		if (!train_in_linear_colors) {
			rgbtarget = linear_to_srgb(rgbtarget);
			background_color = linear_to_srgb(background_color);
		}
	} else if (color_space == EColorSpace::SRGB) {
		background_color = linear_to_srgb(background_color);
		if (texsamp.w() > 0) {
			rgbtarget = linear_to_srgb(exposure_scale * texsamp.head<3>() / texsamp.w()) * texsamp.w() + (1.0f - texsamp.w()) * background_color;
		} else {
			rgbtarget = background_color;
		}
	}

	if (compacted_numsteps == numsteps) {
		// support arbitrary background colors
		rgb_ray += T * background_color;
	}

	// Step again, this time computing loss
	network_output -= 4 * compacted_numsteps; // rewind the pointer
	coords_in -= compacted_numsteps;

	uint32_t compacted_base = atomicAdd(numsteps_counter, compacted_numsteps); // first entry in the array is a counter
	compacted_numsteps = min(max_samples_compacted - min(max_samples_compacted, compacted_base), compacted_numsteps);
	numsteps_in[i*2+0] = compacted_numsteps;
	numsteps_in[i*2+1] = compacted_base;
	if (compacted_numsteps == 0) {
		return;
	}

	max_level_compacted_ptr += compacted_base;
	coords_out += compacted_base;

	dloss_doutput += compacted_base * padded_output_width;

	LossAndGradient lg = loss_and_gradient(rgbtarget, rgb_ray, loss_type);
	lg.loss /= img_pdf * xy_pdf;

	// Note: dividing the gradient by the PDF would cause unbiased loss estimates.
	// Essentially: variance reduction, but otherwise the same optimization.
	// We _dont_ want that. If importance sampling is enabled, we _do_ actually want
	// to change the weighting of the loss function. So don't divide.
	// lg.gradient /= img_pdf * xy_pdf;

	float mean_loss = lg.loss.mean();
	if (loss_output) {
		loss_output[i] = mean_loss / (float)n_rays;
	}

	if (error_map) {
		Vector2f pos = (xy.cwiseProduct(error_map_res.cast<float>()) - Vector2f::Constant(0.5f)).cwiseMax(0.0f).cwiseMin(error_map_res.cast<float>() - Vector2f::Constant(1.0f + 1e-4f));
		Vector2i pos_int = pos.cast<int>();
		auto weight = pos - pos_int.cast<float>();

		Vector2i idx = pos_int.cwiseMin(resolution - Vector2i::Constant(2)).cwiseMax(0);

		auto deposit_val = [&](int x, int y, float val) {
			atomicAdd(&error_map[img * error_map_res.prod() + y * error_map_res.x() + x], val);
		};

		if (sharpness_data && aabb.contains(hitpoint)) {
			Vector2i sharpness_pos = xy.cwiseProduct(sharpness_resolution.cast<float>()).cast<int>().cwiseMax(0).cwiseMin(sharpness_resolution - Vector2i::Constant(1));
			float sharp = sharpness_data[img * sharpness_resolution.prod() + sharpness_pos.y() * sharpness_resolution.x() + sharpness_pos.x()] + 1e-6f;

			// The maximum value of positive floats interpreted in uint format is the same as the maximum value of the floats.
			float grid_sharp = __uint_as_float(atomicMax((uint32_t*)&cascaded_grid_at(hitpoint, sharpness_grid, mip_from_pos(hitpoint)), __float_as_uint(sharp)));
			grid_sharp = fmaxf(sharp, grid_sharp); // atomicMax returns the old value, so compute the new one locally.

			mean_loss *= fmaxf(sharp / grid_sharp, 0.01f);
		}

		deposit_val(idx.x(),   idx.y(),   (1 - weight.x()) * (1 - weight.y()) * mean_loss);
		deposit_val(idx.x()+1, idx.y(),        weight.x()  * (1 - weight.y()) * mean_loss);
		deposit_val(idx.x(),   idx.y()+1, (1 - weight.x()) *      weight.y()  * mean_loss);
		deposit_val(idx.x()+1, idx.y()+1,      weight.x()  *      weight.y()  * mean_loss);
	}

	loss_scale /= n_rays;

	const float output_l2_reg = rgb_activation == ENerfActivation::Exponential ? 1e-4f : 0.0f;
	const float output_l1_reg_density = *mean_density_ptr < NERF_MIN_OPTICAL_THICKNESS() ? 1e-4f : 0.0f;

	// now do it again computing gradients
	Array3f rgb_ray2 = { 0.f,0.f,0.f };
	T = 1.f;
	for (uint32_t j = 0; j < compacted_numsteps; ++j) {
		if (max_level_rand_training) {
			max_level_compacted_ptr[j] = max_level;
		}
		// Compact network inputs
		coords_out[j] = coords_in[j];
		float dt = unwarp_dt(coords_in[j].dt);
		const tcnn::vector_t<tcnn::network_precision_t, 4> local_network_output = *(tcnn::vector_t<tcnn::network_precision_t, 4>*)network_output;
		const Array3f rgb = network_to_rgb(local_network_output, rgb_activation);
		const float density = network_to_density(float(local_network_output[3]), density_activation);
		const float alpha = 1.f - __expf(-density * dt);
		const float weight = alpha * T;
		rgb_ray2 += weight * rgb;
		T *= (1.f - alpha);

		// we know the suffix of this ray compared to where we are up to. note the suffix depends on this step's alpha as suffix = (1-alpha)*(somecolor), so dsuffix/dalpha = -somecolor = -suffix/(1-alpha)
		const Array3f suffix = rgb_ray - rgb_ray2;
		const Array3f dloss_by_drgb = weight * lg.gradient;

		tcnn::vector_t<tcnn::network_precision_t, 4> local_dL_doutput;

		// chain rule to go from dloss/drgb to dloss/dmlp_output
		local_dL_doutput[0] = loss_scale * (dloss_by_drgb.x() * network_to_rgb_derivative(local_network_output[0], rgb_activation) + fmaxf(0.0f, output_l2_reg * (float)local_network_output[0])); // Penalize way too large color values
		local_dL_doutput[1] = loss_scale * (dloss_by_drgb.y() * network_to_rgb_derivative(local_network_output[1], rgb_activation) + fmaxf(0.0f, output_l2_reg * (float)local_network_output[1]));
		local_dL_doutput[2] = loss_scale * (dloss_by_drgb.z() * network_to_rgb_derivative(local_network_output[2], rgb_activation) + fmaxf(0.0f, output_l2_reg * (float)local_network_output[2]));

		float density_derivative = network_to_density_derivative(float(local_network_output[3]), density_activation);
		float dloss_by_dmlp = density_derivative * (
			dt * lg.gradient.matrix().dot((T * rgb - suffix).matrix())
		);

		//static constexpr float mask_supervision_strength = 1.f; // we are already 'leaking' mask information into the nerf via the random bg colors; setting this to eg between 1 and  100 encourages density towards 0 in such regions.
		//dloss_by_dmlp += (texsamp.w()<0.001f) ? mask_supervision_strength * weight : 0.f ;

		local_dL_doutput[3] = loss_scale * dloss_by_dmlp + (float(local_network_output[3]) < 0 ? -output_l1_reg_density : 0.0f);

		*(tcnn::vector_t<tcnn::network_precision_t, 4>*)dloss_doutput = local_dL_doutput;

		dloss_doutput += padded_output_width;
		network_output += 4;
	}

	if (exposure_gradient) {
		// Assume symmetric loss
		Array3f dloss_by_dgt = -lg.gradient / xy_pdf;

		if (!train_in_linear_colors) {
			dloss_by_dgt /= srgb_to_linear_derivative(rgbtarget);
		}

		// 2^exposure * log(2)
		Array3f dloss_by_dexposure = loss_scale * dloss_by_dgt * exposure_scale * 0.6931471805599453f;
		atomicAdd(&exposure_gradient[img].x(), dloss_by_dexposure.x());
		atomicAdd(&exposure_gradient[img].y(), dloss_by_dexposure.y());
		atomicAdd(&exposure_gradient[img].z(), dloss_by_dexposure.z());
	}

	if (compacted_numsteps == numsteps && envmap_gradient) {
		Array3f loss_gradient = lg.gradient;
		if (envmap_loss_type != loss_type) {
			loss_gradient = loss_and_gradient(rgbtarget, rgb_ray, envmap_loss_type).gradient;
		}

		Array3f dloss_by_dbackground = T * loss_gradient;
		if (!train_in_linear_colors) {
			dloss_by_dbackground /= srgb_to_linear_derivative(background_color);
		}

		tcnn::vector_t<tcnn::network_precision_t, 4> dL_denvmap;
		dL_denvmap[0] = loss_scale * dloss_by_dbackground.x();
		dL_denvmap[1] = loss_scale * dloss_by_dbackground.y();
		dL_denvmap[2] = loss_scale * dloss_by_dbackground.z();


		float dloss_by_denvmap_alpha = dloss_by_dbackground.matrix().dot(-pre_envmap_background_color.matrix());

		// dL_denvmap[3] = loss_scale * dloss_by_denvmap_alpha;
		dL_denvmap[3] = (tcnn::network_precision_t)0;

		deposit_envmap_gradient(dL_denvmap, envmap_gradient, envmap_resolution, dir);
	}
}

__global__ void compute_cam_gradient_train_nerf(
	const uint32_t n_rays,
	const uint32_t n_rays_total,
	default_rng_t rng,
	const BoundingBox aabb,
	const uint32_t* __restrict__ rays_counter,
	const Matrix<float, 3, 4>* training_xforms,
	Vector2i resolution,
	bool snap_to_pixel_centers,
	Vector3f* cam_pos_gradient,
	Vector3f* cam_rot_gradient,
	const uint32_t n_training_images,
	const uint32_t* __restrict__ ray_indices_in,
	const Ray* __restrict__ rays_in,
	uint32_t* __restrict__ numsteps_in,
	const NerfCoordinate* __restrict__ coords,
	const NerfCoordinate* __restrict__ coords_gradient,
	float* __restrict__ distortion_gradient,
	float* __restrict__ distortion_gradient_weight,
	const Vector2i distortion_resolution,
	Vector2f* cam_focal_length_gradient,
	const float* __restrict__ cdf_x_cond_y,
	const float* __restrict__ cdf_y,
	const float* __restrict__ cdf_img,
	const Vector2i error_map_res
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= *rays_counter) { return; }

	// grab the number of samples for this ray, and the first sample
	uint32_t numsteps = numsteps_in[i*2+0];
	if (numsteps == 0) {
		// The ray doesn't matter. So no gradient onto the camera
		return;
	}

	uint32_t base = numsteps_in[i*2+1];
	coords += base;
	coords_gradient += base;

	// Must be same seed as above to obtain the same
	// background color.
	uint32_t ray_idx = ray_indices_in[i];
	uint32_t img = image_idx(ray_idx, n_rays, n_rays_total, n_training_images, cdf_img);

	const Matrix<float, 3, 4>& xform = training_xforms[img];

	Ray ray = rays_in[i];
	Ray ray_gradient = { Vector3f::Zero(), Vector3f::Zero() };

	// Compute ray gradient
	for (uint32_t j = 0; j < numsteps; ++j) {
		// pos = ray.o + t * ray.d;

		const Vector3f warped_pos = coords[j].pos.p;
		const Vector3f pos_gradient = coords_gradient[j].pos.p.cwiseProduct(warp_position_derivative(warped_pos, aabb));
		ray_gradient.o += pos_gradient;
		const Vector3f pos = unwarp_position(warped_pos, aabb);

		// Scaled by t to account for the fact that further-away objects' position
		// changes more rapidly as the direction changes.
		float t = (pos - ray.o).norm();
		const Vector3f dir_gradient = coords_gradient[j].dir.d.cwiseProduct(warp_direction_derivative(coords[j].dir.d));
		ray_gradient.d += pos_gradient * t + dir_gradient;
	}

	// Projection of the raydir gradient onto the plane normal to raydir,
	// because that's the only degree of motion that the raydir has.
	ray_gradient.d -= ray.d * ray_gradient.d.dot(ray.d);

	rng.advance(ray_idx * N_MAX_RANDOM_SAMPLES_PER_RAY());
	float xy_pdf = 1.0f;
	Vector2f xy = nerf_random_image_pos_training(rng, resolution, snap_to_pixel_centers, cdf_x_cond_y, cdf_y, error_map_res, img, &xy_pdf);

	if (distortion_gradient) {
		// Rotate ray gradient to obtain image plane gradient.
		// This has the effect of projecting the (already projected) ray gradient from the
		// tangent plane of the sphere onto the image plane (which is correct!).
		Vector3f image_plane_gradient = xform.block<3,3>(0,0).inverse() * ray_gradient.d;

		// Splat the resulting 2D image plane gradient into the distortion params
		deposit_image_gradient<2>(image_plane_gradient.head<2>() / xy_pdf, distortion_gradient, distortion_gradient_weight, distortion_resolution, xy);
	}

	if (cam_pos_gradient) {
		// Atomically reduce the ray gradient into the xform gradient
		#pragma unroll
		for (uint32_t j = 0; j < 3; ++j) {
			atomicAdd(&cam_pos_gradient[img][j], ray_gradient.o[j] / xy_pdf);
		}
	}

	if (cam_rot_gradient) {
		// Rotation is averaged in log-space (i.e. by averaging angle-axes).
		// Due to our construction of ray_gradient.d, ray_gradient.d and ray.d are
		// orthogonal, leading to the angle_axis magnitude to equal the magnitude
		// of ray_gradient.d.
		Vector3f angle_axis = ray.d.cross(ray_gradient.d);

		// Atomically reduce the ray gradient into the xform gradient
		#pragma unroll
		for (uint32_t j = 0; j < 3; ++j) {
			atomicAdd(&cam_rot_gradient[img][j], angle_axis[j] / xy_pdf);
		}
	}
}

__global__ void shade_kernel_nerf(const uint32_t n_elements, Array4f* rgba, NerfPayload* payloads, ERenderMode render_mode, bool train_in_linear_colors, Array4f* frame_buffer) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;
	NerfPayload& payload = payloads[i];

	Array4f tmp = rgba[i];

	if (render_mode == ERenderMode::Normals) {
		Array3f n = tmp.head<3>().matrix().normalized().array();
		tmp.head<3>() = (0.5f * n + Array3f::Constant(0.5f)) * tmp.w();
	} else if (render_mode == ERenderMode::Cost) {
		float col = (float)payload.n_steps / 128;
		tmp = {col, col, col, 1.0f};
	}

	if (!train_in_linear_colors && (render_mode == ERenderMode::Shade || render_mode == ERenderMode::Slice)) {
		// Accumulate in linear colors
		tmp.head<3>() = srgb_to_linear(tmp.head<3>());
	}

	frame_buffer[payload.idx] = tmp + frame_buffer[payload.idx] * (1.0f - tmp.w());
}

__global__ void compact_kernel_nerf(
	const uint32_t n_elements,
	Array4f* src_rgba, NerfPayload* src_payloads,
	Array4f* dst_rgba, NerfPayload* dst_payloads,
	Array4f* dst_final_rgba, NerfPayload* dst_final_payloads,
	uint32_t* counter, uint32_t* finalCounter
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	NerfPayload& src_payload = src_payloads[i];

	if (src_payload.alive) {
		uint32_t idx = atomicAdd(counter, 1);
		dst_payloads[idx] = src_payload;
		dst_rgba[idx] = src_rgba[i];
	} else if (src_rgba[i].w() > 0.001f) {
		uint32_t idx = atomicAdd(finalCounter, 1);
		dst_final_payloads[idx] = src_payload;
		dst_final_rgba[idx] = src_rgba[i];
	}
}

__global__ void init_rays_with_payload_kernel_nerf(
	uint32_t spp,
	NerfPayload* __restrict__ payloads,
	Vector2i resolution,
	Vector2f focal_length,
	Matrix<float, 3, 4> camera_matrix0,
	Matrix<float, 3, 4> camera_matrix1,
	Vector2f screen_center,
	bool snap_to_pixel_centers,
	BoundingBox aabb,
	float plane_z,
	float dof,
	CameraDistortion camera_distortion,
	const float* __restrict__ envmap_data,
	const Vector2i envmap_resolution,
	Array4f* __restrict__ framebuffer,
	const float* __restrict__ distortion_data,
	const Vector2i distortion_resolution,
	ERenderMode render_mode
) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= resolution.x() || y >= resolution.y()) {
		return;
	}

	uint32_t idx = x + resolution.x() * y;

	if (plane_z < 0) {
		dof = 0.0;
	}

	float ray_time = ld_random_val(spp, idx*72239731);
	// Ray ray = pixel_to_ray_orthographic(
	// 	spp,
	// 	{x, y},
	// 	resolution,
	// 	focal_length,
	// 	camera_matrix0 * ray_time + camera_matrix1 * (1.f - ray_time),
	// 	screen_center,
	// 	plane_z,
	// 	dof
	// );
	Ray ray = pixel_to_ray(
		spp,
		{x, y},
		resolution,
		focal_length,
		camera_matrix0 * ray_time + camera_matrix1 * (1.f - ray_time),
		screen_center,
		snap_to_pixel_centers,
		plane_z,
		dof,
		camera_distortion,
		distortion_data,
		distortion_resolution
	);

	if (plane_z < 0) {
		float n = ray.d.norm();
		NerfPayload& payload = payloads[idx];
		payload.origin = ray.o;
		payload.dir = (1.0f/n) * ray.d;
		payload.t = -plane_z*n;
		payload.idx = idx;
		payload.n_steps = 0;
		payload.alive = false;
		return;
	}

	ray.d = ray.d.normalized();

	if (envmap_data) {
		framebuffer[idx] = read_envmap(envmap_data, envmap_resolution, ray.d);
	}

	float t = fmaxf(aabb.ray_intersect(ray.o, ray.d).x(), NERF_RENDERING_NEAR_DISTANCE()) + 1e-6f;

	NerfPayload& payload = payloads[idx];
	if (!aabb.contains(ray.o + ray.d * t)) {
		payload.origin = ray.o;
		payload.alive = false;
		return;
	}

	if (render_mode == ERenderMode::Distortion) {
		if (distortion_data) {
			framebuffer[idx].head<2>() = read_image<2>(distortion_data, distortion_resolution, Vector2f((float)x + 0.5f, (float)y + 0.5f).cwiseQuotient(resolution.cast<float>())) * 50.0f + Vector2f::Constant(0.5f);
		} else {
			framebuffer[idx].head<2>() = Vector2f{0.5f, 0.5f};
		}
		framebuffer[idx].tail<2>() = Vector2f{0.5f, 1.0f};
		payload.origin = ray.o + ray.d * 10000.0f;
		payload.alive = false;
		return;
	}

	payload.origin = ray.o;
	payload.dir = ray.d;
	payload.t = t;
	payload.idx = idx;
	payload.n_steps = 0;
	payload.alive = true;
}

static constexpr float MIN_PDF = 0.01f;

__global__ void construct_cdf_2d(
	uint32_t n_images,
	uint32_t height,
	uint32_t width,
	const float* __restrict__ data,
	float* __restrict__ cdf_x_cond_y,
	float* __restrict__ cdf_y
) {
	const uint32_t y = threadIdx.x + blockIdx.x * blockDim.x;
	const uint32_t img = threadIdx.y + blockIdx.y * blockDim.y;
	if (y >= height || img >= n_images) return;

	const uint32_t offset_xy = img * height * width + y * width;
	data += offset_xy;
	cdf_x_cond_y += offset_xy;

	float cum = 0;
	for (uint32_t x = 0; x < width; ++x) {
		cum += data[x] + 1e-10f;
		cdf_x_cond_y[x] = cum;
	}

	cdf_y[img * height + y] = cum;
	float norm = __frcp_rn(cum);

	for (uint32_t x = 0; x < width; ++x) {
		cdf_x_cond_y[x] = (1.0f - MIN_PDF) * cdf_x_cond_y[x] * norm + MIN_PDF * (float)(x+1) / (float)width;
	}
}

__global__ void construct_cdf_1d(
	uint32_t n_images,
	uint32_t height,
	float* __restrict__ cdf_y,
	float* __restrict__ cdf_img
) {
	const uint32_t img = threadIdx.x + blockIdx.x * blockDim.x;
	if (img >= n_images) return;

	cdf_y += img * height;

	float cum = 0;
	for (uint32_t y = 0; y < height; ++y) {
		cum += cdf_y[y];
		cdf_y[y] = cum;
	}

	cdf_img[img] = cum;

	float norm = __frcp_rn(cum);
	for (uint32_t y = 0; y < height; ++y) {
		cdf_y[y] = (1.0f - MIN_PDF) * cdf_y[y] * norm + MIN_PDF * (float)(y+1) / (float)height;
	}
}

__global__ void safe_divide(const uint32_t num_elements, float* __restrict__ inout, const float* __restrict__ divisor) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= num_elements) return;

	float local_divisor = divisor[i];
	inout[i] = local_divisor > 0.0f ? (inout[i] / local_divisor) : 0.0f;
}

void Testbed::NerfTracer::init_rays_from_camera(
	uint32_t spp,
	uint32_t padded_output_width,
	const Vector2i& resolution,
	const Vector2f& focal_length,
	const Matrix<float, 3, 4>& camera_matrix0,
	const Matrix<float, 3, 4>& camera_matrix1,
	Vector2f screen_center,
	bool snap_to_pixel_centers,
	const BoundingBox& render_aabb,
	float plane_z,
	float dof,
	const CameraDistortion& camera_distortion,
	const float* envmap_data,
	const Vector2i& envmap_resolution,
	const float* distortion_data,
	const Vector2i& distortion_resolution,
	Eigen::Array4f* frame_buffer,
	uint8_t *grid,
	int show_accel,
	float cone_angle_constant,
	ERenderMode render_mode,
	hipStream_t stream
) {
	// Make sure we have enough memory reserved to render at the requested resolution
	size_t n_pixels = (size_t)resolution.x() * resolution.y();
	enlarge(n_pixels, padded_output_width, stream);

	const dim3 threads = { 16, 8, 1 };
	const dim3 blocks = { div_round_up((uint32_t)resolution.x(), threads.x), div_round_up((uint32_t)resolution.y(), threads.y), 1 };
	init_rays_with_payload_kernel_nerf<<<blocks, threads, 0, stream>>>(
		spp,
		m_rays[0].payload,
		resolution,
		focal_length,
		camera_matrix0,
		camera_matrix1,
		screen_center,
		snap_to_pixel_centers,
		render_aabb,
		plane_z,
		dof,
		camera_distortion,
		envmap_data,
		envmap_resolution,
		frame_buffer,
		distortion_data,
		distortion_resolution,
		render_mode
	);

	m_n_rays_initialized = resolution.x() * resolution.y();

	CUDA_CHECK_THROW(hipMemsetAsync(m_rays[0].rgba, 0, m_n_rays_initialized * sizeof(Array4f), stream));

	linear_kernel(advance_pos_nerf, 0, stream,
		m_n_rays_initialized,
		render_aabb,
		camera_matrix1.col(2),
		focal_length,
		spp,
		m_rays[0].payload,
		grid,
		(show_accel >= 0) ? show_accel : 0,
		cone_angle_constant
	);
}

uint32_t Testbed::NerfTracer::trace(
	NerfNetwork<network_precision_t>& network,
	const BoundingBox& render_aabb,
	const BoundingBox& train_aabb,
	const uint32_t n_training_images,
	const Eigen::Matrix<float, 3, 4>* training_xforms,
	const Vector2f& focal_length,
	float cone_angle_constant,
	const uint8_t* grid,
	ERenderMode render_mode,
	const Eigen::Matrix<float, 3, 4> &camera_matrix,
	float depth_scale,
	int visualized_layer,
	int visualized_dim,
	ENerfActivation rgb_activation,
	ENerfActivation density_activation,
	int show_accel,
	float min_alpha,
	hipStream_t stream
) {
	if (m_n_rays_initialized == 0) {
		return 0;
	}

	CUDA_CHECK_THROW(hipMemsetAsync(m_hit_counter.data(), 0, sizeof(uint32_t), stream));

	uint32_t n_alive = m_n_rays_initialized;
	// m_n_rays_initialized = 0;

	uint32_t i = 1;
	uint32_t double_buffer_index = 0;
	while (i < MARCH_ITER) {
		RaysNerfSoa& rays_current = m_rays[(double_buffer_index + 1) % 2];
		RaysNerfSoa& rays_tmp = m_rays[double_buffer_index % 2];
		++double_buffer_index;

		// Compact rays that did not diverge yet
		{
			CUDA_CHECK_THROW(hipMemsetAsync(m_alive_counter.data(), 0, sizeof(uint32_t), stream));
			linear_kernel(compact_kernel_nerf, 0, stream,
				n_alive,
				rays_tmp.rgba, rays_tmp.payload,
				rays_current.rgba, rays_current.payload,
				m_rays_hit.rgba, m_rays_hit.payload,
				m_alive_counter.data(), m_hit_counter.data()
			);
			CUDA_CHECK_THROW(hipMemcpyAsync(&n_alive, m_alive_counter.data(), sizeof(uint32_t), hipMemcpyDeviceToHost, stream));
			CUDA_CHECK_THROW(hipStreamSynchronize(stream));
		}

		if (n_alive == 0) {
			break;
		}

		uint32_t n_steps_between_compaction = tcnn::clamp(m_n_rays_initialized / n_alive, (uint32_t)MIN_STEPS_INBETWEEN_COMPACTION, (uint32_t)MAX_STEPS_INBETWEEN_COMPACTION);

		linear_kernel(generate_next_nerf_network_inputs, 0, stream,
			n_alive,
			render_aabb,
			train_aabb,
			focal_length,
			camera_matrix.col(2),
			rays_current.payload,
			m_network_input,
			n_steps_between_compaction,
			grid,
			(show_accel>=0) ? show_accel : 0,
			cone_angle_constant
		);
		uint32_t n_elements = next_multiple(n_alive*n_steps_between_compaction, BATCH_SIZE_MULTIPLE);
		GPUMatrix<float> positions_matrix((float*)m_network_input, sizeof(NerfCoordinate)/sizeof(float), n_elements);
		GPUMatrix<network_precision_t> rgbsigma_matrix((network_precision_t*)m_network_output, network.padded_output_width(), n_elements);
		network.inference_mixed_precision(stream, positions_matrix, rgbsigma_matrix);

		if (render_mode == ERenderMode::Normals) {
			network.input_gradient(stream, 3, positions_matrix, positions_matrix);
		} else if (render_mode == ERenderMode::EncodingVis) {
			network.visualize_activation(stream, visualized_layer, visualized_dim, positions_matrix, positions_matrix);
		}

		linear_kernel(composite_kernel_nerf, 0, stream,
			n_alive,
			i,
			train_aabb,
			n_training_images,
			training_xforms,
			camera_matrix,
			focal_length,
			depth_scale,
			rays_current.rgba,
			rays_current.payload,
			m_network_input,
			m_network_output,
			network.padded_output_width(),
			n_steps_between_compaction,
			render_mode,
			grid,
			rgb_activation,
			density_activation,
			show_accel,
			min_alpha
		);

		i += n_steps_between_compaction;
	}

	uint32_t n_hit;
	CUDA_CHECK_THROW(hipMemcpyAsync(&n_hit, m_hit_counter.data(), sizeof(uint32_t), hipMemcpyDeviceToHost, stream));
	CUDA_CHECK_THROW(hipStreamSynchronize(stream));
	return n_hit;
}

void Testbed::NerfTracer::enlarge(size_t n_elements, uint32_t padded_output_width, hipStream_t stream) {
	n_elements = next_multiple(n_elements, size_t(BATCH_SIZE_MULTIPLE)); // network inference rounds n_elements up to 256, and uses these arrays, so we must do so also.

	auto scratch = allocate_workspace_and_distribute<
		Array4f, NerfPayload, // m_rays[0]
		Array4f, NerfPayload, // m_rays[1]
		Array4f, NerfPayload, // m_rays_hit

		network_precision_t,
		NerfCoordinate
	>(
		stream, &m_scratch_alloc,
		n_elements, n_elements,
		n_elements, n_elements,
		n_elements, n_elements,
		n_elements * MAX_STEPS_INBETWEEN_COMPACTION * padded_output_width,
		n_elements * MAX_STEPS_INBETWEEN_COMPACTION
	);

	m_rays[0].set(std::get<0>(scratch), std::get<1>(scratch), n_elements);
	m_rays[1].set(std::get<2>(scratch), std::get<3>(scratch), n_elements);
	m_rays_hit.set(std::get<4>(scratch), std::get<5>(scratch), n_elements);

	m_network_output = std::get<6>(scratch);
	m_network_input = std::get<7>(scratch);
}

void Testbed::render_nerf(CudaRenderBuffer& render_buffer, const Vector2i& max_res, const Vector2f& focal_length, const Matrix<float, 3, 4>& camera_matrix0, const Matrix<float, 3, 4>& camera_matrix1, const Vector2f& screen_center, hipStream_t stream) {
	float plane_z = m_slice_plane_z + m_scale;
	if (m_render_mode == ERenderMode::Slice) {
		plane_z = -plane_z;
	}

	ERenderMode render_mode = m_visualized_dimension > -1 ? ERenderMode::EncodingVis : m_render_mode;

	ScopeGuard tmp_memory_guard{[&]() {
		m_nerf.tracer.clear();
	}};

	m_nerf.tracer.init_rays_from_camera(
		render_buffer.spp(),
		m_network->padded_output_width(),
		render_buffer.resolution(),
		focal_length,
		camera_matrix0,
		camera_matrix1,
		screen_center,
		m_snap_to_pixel_centers,
		m_render_aabb,
		plane_z,
		m_dof,
		m_nerf.render_with_camera_distortion ? m_nerf.training.dataset.camera_distortion : CameraDistortion{},
		m_envmap.envmap->params_inference(),
		m_envmap.resolution,
		m_nerf.render_with_camera_distortion ? m_distortion.map->params_inference() : nullptr,
		m_distortion.resolution,
		render_buffer.frame_buffer(),
		m_nerf.density_grid_bitfield.data(),
		m_nerf.show_accel,
		m_nerf.cone_angle_constant,
		render_mode,
		stream
	);

	uint32_t n_hit;
	if (m_render_mode == ERenderMode::Slice) {
		n_hit = m_nerf.tracer.n_rays_initialized();
	} else {
		float depth_scale = 1.f/m_nerf.training.dataset.scale;
		n_hit = m_nerf.tracer.trace(
			*m_nerf_network,
			m_render_aabb,
			m_aabb,
			m_nerf.training.n_images,
			m_nerf.training.transforms.data(),
			focal_length,
			m_nerf.cone_angle_constant,
			m_nerf.density_grid_bitfield.data(),
			render_mode, camera_matrix1, depth_scale, m_visualized_layer, m_visualized_dimension,
			m_nerf.rgb_activation, m_nerf.density_activation, m_nerf.show_accel, m_nerf.rendering_min_alpha,
			stream
		);
	}
	RaysNerfSoa& rays_hit = m_render_mode == ERenderMode::Slice ? m_nerf.tracer.rays_init() : m_nerf.tracer.rays_hit();

	if (m_render_mode == ERenderMode::Slice) {
		// Store colors in the normal buffer
		uint32_t n_elements = next_multiple(n_hit, BATCH_SIZE_MULTIPLE);

		m_nerf.vis_input.enlarge(n_elements);
		m_nerf.vis_rgba.enlarge(n_elements);
		linear_kernel(generate_nerf_network_inputs_at_current_position, 0, stream, n_hit, m_aabb, rays_hit.payload, m_nerf.vis_input.data());

		GPUMatrix<float> positions_matrix((float*)m_nerf.vis_input.data(), sizeof(NerfCoordinate)/sizeof(float), n_elements);
		GPUMatrix<float> rgbsigma_matrix((float*)m_nerf.vis_rgba.data(), 4, n_elements);

		if (m_visualized_dimension == -1) {
			m_network->inference(stream, positions_matrix, rgbsigma_matrix);
			linear_kernel(compute_nerf_density, 0, stream, n_hit, m_nerf.vis_rgba.data(), m_nerf.rgb_activation, m_nerf.density_activation);
		} else {
			m_network->visualize_activation(stream, m_visualized_layer, m_visualized_dimension, positions_matrix, rgbsigma_matrix);
		}

		linear_kernel(shade_kernel_nerf, 0, stream,
			n_hit,
			m_nerf.vis_rgba.data(),
			rays_hit.payload,
			m_render_mode,
			m_nerf.training.linear_colors,
			render_buffer.frame_buffer()
		);
		return;
	}

	linear_kernel(shade_kernel_nerf, 0, stream,
		n_hit,
		rays_hit.rgba,
		rays_hit.payload,
		m_render_mode,
		m_nerf.training.linear_colors,
		render_buffer.frame_buffer()
	);

	if (render_mode == ERenderMode::Cost) {
		std::vector<NerfPayload> payloads_final_cpu(n_hit);
		CUDA_CHECK_THROW(hipMemcpyAsync(payloads_final_cpu.data(), rays_hit.payload, n_hit * sizeof(NerfPayload), hipMemcpyDeviceToHost, stream));
		CUDA_CHECK_THROW(hipStreamSynchronize(stream));

		size_t total_n_steps = 0;
		for (uint32_t i = 0; i < n_hit; ++i) {
			total_n_steps += payloads_final_cpu[i].n_steps;
		}
		tlog::info() << "Total steps per hit= " << total_n_steps << "/" << n_hit << " = " << ((float)total_n_steps/(float)n_hit);
	}
}

void Testbed::update_nerf_focal_lengths() {
	std::vector<Vector2f> updated_focal_lengths;
	updated_focal_lengths.reserve(m_nerf.training.n_images);
	for (uint32_t i = 0; i < m_nerf.training.n_images; ++i) {
		updated_focal_lengths.push_back(m_nerf.training.dataset.focal_lengths[i] + m_nerf.training.cam_focal_length_offset.variable());
	}
	m_nerf.training.focal_lengths_gpu.resize_and_copy_from_host(updated_focal_lengths);
}

void Testbed::update_nerf_transforms() {
	m_nerf.training.transforms.resize(m_nerf.training.n_images);
	for (uint32_t i = 0; i < m_nerf.training.n_images; ++i) {
		auto xform = m_nerf.training.dataset.xforms[i];

		Vector3f rot = m_nerf.training.cam_rot_offset[i].variable();
		float angle = rot.norm();
		rot /= angle;
		if (angle > 0) {
			xform.block<3,3>(0,0) = AngleAxisf(angle, rot) * xform.block<3,3>(0,0);
		}

		xform.col(3) += m_nerf.training.cam_pos_offset[i].variable();

		m_nerf.training.transforms[i] = xform;
	}

	m_nerf.training.transforms_gpu.resize_and_copy_from_host(m_nerf.training.transforms);
}

void Testbed::load_nerf() {
	if (!m_data_path.empty()) {
		std::vector<fs::path> json_paths;
		if (m_data_path.is_directory()) {
			for (const auto& path : fs::directory{m_data_path}) {
				if (path.is_file() && equals_case_insensitive(path.extension(), "json")) {
					json_paths.emplace_back(path);
				}
			}
		} else if (equals_case_insensitive(m_data_path.extension(), "msgpack")) {
			load_snapshot(m_data_path.str());
			set_train(false);
			return;
		} else if (equals_case_insensitive(m_data_path.extension(), "json")) {
			json_paths.emplace_back(m_data_path);
		} else {
			throw std::runtime_error{"NeRF data path must either be a json file or a directory containing json files."};
		}

		m_nerf.training.dataset = ngp::load_nerf(json_paths, m_nerf.sharpen);
	}

	m_nerf.rgb_activation = m_nerf.training.dataset.is_hdr ? ENerfActivation::Exponential : ENerfActivation::Logistic;

	m_nerf.training.image_resolution = m_nerf.training.dataset.image_resolution;
	m_nerf.training.n_images = (int)m_nerf.training.dataset.n_images;

	m_nerf.training.focal_lengths = m_nerf.training.dataset.focal_lengths;
	m_nerf.training.focal_lengths_gpu.resize_and_copy_from_host(m_nerf.training.focal_lengths);

	m_nerf.training.cam_pos_gradient.resize(m_nerf.training.n_images, Vector3f::Zero());
	m_nerf.training.cam_pos_gradient_gpu.resize_and_copy_from_host(m_nerf.training.cam_pos_gradient);

	m_nerf.training.cam_pos_offset.resize(m_nerf.training.n_images, AdamOptimizer<Vector3f>(1e-4f));
	m_nerf.training.cam_rot_offset.resize(m_nerf.training.n_images, RotationAdamOptimizer(1e-4f));
	m_nerf.training.cam_focal_length_offset = AdamOptimizer<Vector2f>(1e-4f);

	m_nerf.training.cam_rot_gradient.resize(m_nerf.training.n_images, Vector3f::Zero());
	m_nerf.training.cam_rot_gradient_gpu.resize_and_copy_from_host(m_nerf.training.cam_rot_gradient);

	m_nerf.training.cam_exposure_gradient.resize(m_nerf.training.n_images, Array3f::Zero());
	m_nerf.training.cam_exposure_gpu.resize_and_copy_from_host(m_nerf.training.cam_exposure_gradient);
	m_nerf.training.cam_exposure_gradient_gpu.resize_and_copy_from_host(m_nerf.training.cam_exposure_gradient);

	m_nerf.training.cam_focal_length_gradient = Vector2f::Zero();
	m_nerf.training.cam_focal_length_gradient_gpu.resize_and_copy_from_host(&m_nerf.training.cam_focal_length_gradient, 1);

	if (m_nerf.training.dataset.rays_data.data()) {
		m_nerf.training.near_distance = 0.0f;
		// m_nerf.training.optimize_exposure = true;
	}

	// Uncomment the following line to see how the network learns distortion from scratch rather than
	// starting from the distortion that's described by the training data.
	// m_nerf.training.dataset.camera_distortion = {};

	// Perturbation of the training cameras -- for debugging the online extrinsics learning code
	// for (uint32_t i = 0; i < m_nerf.training.n_images; ++i) {
	// 	Vector3f rot = random_val_3d(i+1) * 0.01f;
	// 	float angle = rot.norm();
	// 	rot /= angle;
	// 	m_nerf.training.dataset.xforms[i].block<3,3>(0,0) = AngleAxisf(angle, rot).matrix() * m_nerf.training.dataset.xforms[i].block<3,3>(0,0);
	// 	m_nerf.training.dataset.xforms[i].col(3) += random_val_3d(i+1+m_nerf.training.n_images) * 0.01f;
	// }

	update_nerf_transforms();

	m_aabb = BoundingBox{Vector3f::Constant(0.5f), Vector3f::Constant(0.5f)};
	m_aabb.inflate(0.5f * std::min(1 << (NERF_CASCADES()-1), m_nerf.training.dataset.aabb_scale));
	m_raw_aabb = m_aabb;
	m_render_aabb = m_aabb;
	if (!m_nerf.training.dataset.render_aabb.is_empty()) {
		m_render_aabb = m_nerf.training.dataset.render_aabb.intersection(m_aabb);
	}

	m_nerf.max_cascade = 0;
	while ((1 << m_nerf.max_cascade) < m_nerf.training.dataset.aabb_scale) {
		++m_nerf.max_cascade;
	}

	// Perform fixed-size stepping in unit-cube scenes (like original NeRF) and exponential
	// stepping in larger scenes.
	m_nerf.cone_angle_constant = m_nerf.training.dataset.aabb_scale <= 1 ? 0.0f : (1.0f / 256.0f);

	m_up_dir = m_nerf.training.dataset.up;
}

void Testbed::update_density_grid_nerf(float decay, uint32_t n_uniform_density_grid_samples, uint32_t n_nonuniform_density_grid_samples, hipStream_t stream) {
	const uint32_t n_elements = NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_CASCADES();

	m_nerf.density_grid.enlarge(n_elements);

	const uint32_t n_density_grid_samples = n_uniform_density_grid_samples + n_nonuniform_density_grid_samples;

	const uint32_t padded_output_width = m_nerf_network->padded_density_output_width();

	GPUMemoryArena::Allocation alloc;
	auto scratch = allocate_workspace_and_distribute<
		NerfPosition,       // positions at which the NN will be queried for density evaluation
		uint32_t,           // indices of corresponding density grid cells
		float,              // the resulting densities `density_grid_tmp` to be merged with the running estimate of the grid
		network_precision_t // output of the MLP before being converted to densities.
	>(stream, &alloc, n_density_grid_samples, n_elements, n_elements, n_density_grid_samples * padded_output_width);

	NerfPosition* density_grid_positions = std::get<0>(scratch);
	uint32_t* density_grid_indices = std::get<1>(scratch);
	float* density_grid_tmp = std::get<2>(scratch);
	network_precision_t* mlp_out = std::get<3>(scratch);

	if (m_training_step == 0) {
		m_nerf.density_grid_ema_step = 0;
		// Only cull away empty regions where no camera is looking when the cameras are actually meaningful.
		if (!m_nerf.training.dataset.rays_data.data()) {
			linear_kernel(mark_untrained_density_grid, 0, stream, n_elements, m_nerf.density_grid.data(),
				m_nerf.training.n_images,
				m_nerf.training.focal_lengths_gpu.data(),
				m_nerf.training.transforms_gpu.data(),
				m_nerf.training.image_resolution
			);
		} else {
			CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.density_grid.data(), 0, sizeof(float)*n_elements, stream));
		}
	}

	uint32_t n_steps = 1;
	for (uint32_t i = 0; i < n_steps; ++i) {
		CUDA_CHECK_THROW(hipMemsetAsync(density_grid_tmp, 0, sizeof(float)*n_elements, stream));

		linear_kernel(generate_grid_samples_nerf_nonuniform, 0, stream,
			n_uniform_density_grid_samples,
			m_rng,
			m_nerf.density_grid_ema_step,
			m_aabb,
			m_nerf.density_grid.data(),
			density_grid_positions,
			density_grid_indices,
			m_nerf.max_cascade+1,
			-0.01f
		);
		m_rng.advance();

		linear_kernel(generate_grid_samples_nerf_nonuniform, 0, stream,
			n_nonuniform_density_grid_samples,
			m_rng,
			m_nerf.density_grid_ema_step,
			m_aabb,
			m_nerf.density_grid.data(),
			density_grid_positions+n_uniform_density_grid_samples,
			density_grid_indices+n_uniform_density_grid_samples,
			m_nerf.max_cascade+1,
			NERF_MIN_OPTICAL_THICKNESS()
		);
		m_rng.advance();

		GPUMatrix<network_precision_t> rgbsigma_matrix(mlp_out, padded_output_width, n_density_grid_samples);
		m_nerf_network->density(stream, {(float*)density_grid_positions, sizeof(NerfPosition)/sizeof(float)}, rgbsigma_matrix, false);

		linear_kernel(splat_grid_samples_nerf_max_nearest_neighbor, 0, stream, n_density_grid_samples, density_grid_indices, padded_output_width, mlp_out, density_grid_tmp, m_nerf.rgb_activation, m_nerf.density_activation);
		linear_kernel(ema_grid_samples_nerf, 0, stream, n_elements, decay, m_nerf.density_grid_ema_step, m_nerf.density_grid.data(), density_grid_tmp);

		++m_nerf.density_grid_ema_step;
	}

	update_density_grid_mean_and_bitfield(stream);
}

void Testbed::update_density_grid_mean_and_bitfield(hipStream_t stream) {
	const uint32_t n_elements = NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_GRIDSIZE();

	size_t size_including_mips = grid_mip_offset(NERF_CASCADES())/8;
	m_nerf.density_grid_bitfield.enlarge(size_including_mips);
	m_nerf.density_grid_mean.enlarge(reduce_sum_workspace_size(n_elements));

	CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.density_grid_mean.data(), 0, sizeof(float), stream));
	reduce_sum(m_nerf.density_grid.data(), [n_elements] __device__ (float val) { return fmaxf(val, 0.f) / (n_elements); }, m_nerf.density_grid_mean.data(), n_elements, stream);

	linear_kernel(grid_to_bitfield, 0, stream, n_elements/8 * NERF_CASCADES(), m_nerf.density_grid.data(), m_nerf.density_grid_bitfield.data(), m_nerf.density_grid_mean.data());

	for (uint32_t level = 1; level < NERF_CASCADES(); ++level) {
		linear_kernel(bitfield_max_pool, 0, stream, n_elements/64, m_nerf.get_density_grid_bitfield_mip(level-1), m_nerf.get_density_grid_bitfield_mip(level));
	}
}

void Testbed::train_nerf(uint32_t target_batch_size, uint32_t n_training_steps, hipStream_t stream) {
	if (m_nerf.training.include_sharpness_in_error) {
		size_t n_cells = NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_CASCADES();
		if (m_nerf.training.sharpness_grid.size() < n_cells) {
			m_nerf.training.sharpness_grid.enlarge(NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_CASCADES());
			CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.sharpness_grid.data(), 0, m_nerf.training.sharpness_grid.get_bytes(), stream));
		}

		if (m_training_step == 0) {
			CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.sharpness_grid.data(), 0, m_nerf.training.sharpness_grid.get_bytes(), stream));
		} else {
			linear_kernel(decay_sharpness_grid_nerf, 0, stream, m_nerf.training.sharpness_grid.size(), 0.95f, m_nerf.training.sharpness_grid.data());
		}
	}

	// number of steps each ray took, and the first offset of samples
	m_nerf.training.numsteps_counter.enlarge(n_training_steps);
	m_nerf.training.numsteps_counter_compacted.enlarge(n_training_steps);

	m_nerf.training.loss.enlarge(m_nerf.training.rays_per_batch * n_training_steps);

	uint32_t* counter = m_nerf.training.numsteps_counter.data();
	uint32_t* compacted_counter = m_nerf.training.numsteps_counter_compacted.data();

	CUDA_CHECK_THROW(hipMemsetAsync(counter, 0, sizeof(uint32_t)*n_training_steps, stream)); // clear the counter in the first slot
	CUDA_CHECK_THROW(hipMemsetAsync(compacted_counter, 0, sizeof(uint32_t)*n_training_steps, stream)); // clear the counter in the first slot
	CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.loss.data(), 0, sizeof(float)*m_nerf.training.rays_per_batch*n_training_steps, stream));

	if (m_nerf.training.n_steps_since_cam_update == 0) {
		CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.cam_pos_gradient_gpu.data(), 0, m_nerf.training.cam_pos_gradient_gpu.get_bytes(), stream));
		CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.cam_rot_gradient_gpu.data(), 0, m_nerf.training.cam_rot_gradient_gpu.get_bytes(), stream));
		CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.cam_exposure_gradient_gpu.data(), 0, m_nerf.training.cam_exposure_gradient_gpu.get_bytes(), stream));
		CUDA_CHECK_THROW(hipMemsetAsync(m_distortion.map->gradients(), 0, sizeof(float)*m_distortion.map->n_params(), stream));
		CUDA_CHECK_THROW(hipMemsetAsync(m_distortion.map->gradient_weights(), 0, sizeof(float)*m_distortion.map->n_params(), stream));
		CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.cam_focal_length_gradient_gpu.data(), 0, m_nerf.training.cam_focal_length_gradient_gpu.get_bytes(), stream));
	}

	if (m_nerf.training.n_steps_since_error_map_update == 0) {
		uint32_t n_samples_per_image = (m_nerf.training.n_steps_between_error_map_updates * m_nerf.training.rays_per_batch) / m_nerf.training.dataset.n_images;
		m_nerf.training.error_map.resolution = Vector2i::Constant((int)(std::sqrt(std::sqrt((float)n_samples_per_image)) * 3.5f)).cwiseMin(m_nerf.training.image_resolution);
		m_nerf.training.error_map.data.resize(m_nerf.training.error_map.resolution.prod() * m_nerf.training.dataset.n_images);
		CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.error_map.data.data(), 0, m_nerf.training.error_map.data.get_bytes(), stream));
	}

	float* envmap_gradient = m_nerf.training.train_envmap ? m_envmap.envmap->gradients() : nullptr;
	if (envmap_gradient) {
		CUDA_CHECK_THROW(hipMemsetAsync(envmap_gradient, 0, sizeof(float)*m_envmap.envmap->n_params(), stream));
	}

	for (uint32_t i = 0; i < n_training_steps; ++i) {
		train_nerf_step(target_batch_size, m_nerf.training.rays_per_batch, counter+i, compacted_counter+i, m_nerf.training.loss.data()+i*m_nerf.training.rays_per_batch, m_training_stream);
	}

	if (envmap_gradient) {
		m_envmap.trainer->optimizer_step(stream, LOSS_SCALE*(float)n_training_steps);
	}

	std::vector<uint32_t> counter_cpu(n_training_steps);
	std::vector<uint32_t> compacted_counter_cpu(n_training_steps);
	m_nerf.training.numsteps_counter.copy_to_host(counter_cpu, n_training_steps);
	m_nerf.training.numsteps_counter_compacted.copy_to_host(compacted_counter_cpu, n_training_steps);

	m_nerf.training.measured_batch_size = 0;
	m_nerf.training.measured_batch_size_before_compaction = 0;

	for (uint32_t i = 0; i < n_training_steps; ++i) {
		if (counter_cpu[i] == 0 || compacted_counter_cpu[i] == 0) {
			m_train = false;
			tlog::warning() << "Nerf training generated 0 samples. Aborting training.";
			return;
		}

		m_nerf.training.measured_batch_size_before_compaction += counter_cpu[i];
		m_nerf.training.measured_batch_size += compacted_counter_cpu[i];
	}

	m_nerf.training.measured_batch_size_before_compaction /= n_training_steps;
	m_nerf.training.measured_batch_size /= n_training_steps;

	m_loss_scalar = reduce_sum(m_nerf.training.loss.data(), m_nerf.training.rays_per_batch * n_training_steps, stream) / (float)(n_training_steps);
	m_loss_scalar *= (float)m_nerf.training.measured_batch_size / (float)target_batch_size;
	update_loss_graph();

	m_nerf.training.rays_per_batch = (uint32_t)((float)m_nerf.training.rays_per_batch * (float)target_batch_size / (float)m_nerf.training.measured_batch_size);
	m_nerf.training.rays_per_batch = std::min(next_multiple(m_nerf.training.rays_per_batch, BATCH_SIZE_MULTIPLE), 1u << 18);

	// Compute CDFs from the error map
	m_nerf.training.n_steps_since_error_map_update += n_training_steps;
	// This is low-overhead enough to warrant always being on.
	// It makes for useful visualizations of the training error.
	bool accumulate_error = true;
	if (accumulate_error && m_nerf.training.n_steps_since_error_map_update >= m_nerf.training.n_steps_between_error_map_updates) {
		m_nerf.training.error_map.cdf_resolution = m_nerf.training.error_map.resolution;
		m_nerf.training.error_map.cdf_x_cond_y.resize(m_nerf.training.error_map.cdf_resolution.prod() * m_nerf.training.dataset.n_images);
		m_nerf.training.error_map.cdf_y.resize(m_nerf.training.error_map.cdf_resolution.y() * m_nerf.training.dataset.n_images);
		m_nerf.training.error_map.cdf_img.resize(m_nerf.training.dataset.n_images);

		CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.error_map.cdf_x_cond_y.data(), 0, m_nerf.training.error_map.cdf_x_cond_y.get_bytes(), stream));
		CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.error_map.cdf_y.data(), 0, m_nerf.training.error_map.cdf_y.get_bytes(), stream));
		CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.error_map.cdf_img.data(), 0, m_nerf.training.error_map.cdf_img.get_bytes(), stream));

		const dim3 threads = { 16, 8, 1 };
		const dim3 blocks = { div_round_up((uint32_t)m_nerf.training.error_map.cdf_resolution.y(), threads.x), div_round_up((uint32_t)m_nerf.training.dataset.n_images, threads.y), 1 };
		construct_cdf_2d<<<blocks, threads, 0, stream>>>(
			m_nerf.training.dataset.n_images, m_nerf.training.error_map.cdf_resolution.y(), m_nerf.training.error_map.cdf_resolution.x(),
			m_nerf.training.error_map.data.data(),
			m_nerf.training.error_map.cdf_x_cond_y.data(),
			m_nerf.training.error_map.cdf_y.data()
		);
		linear_kernel(construct_cdf_1d, 0, stream,
			m_nerf.training.dataset.n_images,
			m_nerf.training.error_map.cdf_resolution.y(),
			m_nerf.training.error_map.cdf_y.data(),
			m_nerf.training.error_map.cdf_img.data()
		);

		// Compute image CDF on the CPU. It's single-threaded anyway. No use parallelizing.
		m_nerf.training.error_map.pmf_img_cpu.resize(m_nerf.training.error_map.cdf_img.size());
		m_nerf.training.error_map.cdf_img.copy_to_host(m_nerf.training.error_map.pmf_img_cpu);
		std::vector<float> cdf_img_cpu = m_nerf.training.error_map.pmf_img_cpu; // Copy unnormalized PDF into CDF buffer
		float cum = 0;
		for (float& f : cdf_img_cpu) {
			cum += f;
			f = cum;
		}
		float norm = 1.0f / cum;
		for (size_t i = 0; i < cdf_img_cpu.size(); ++i) {
			constexpr float MIN_PMF = 0.1f;
			m_nerf.training.error_map.pmf_img_cpu[i] = (1.0f - MIN_PMF) * m_nerf.training.error_map.pmf_img_cpu[i] * norm + MIN_PMF / (float)m_nerf.training.dataset.n_images;
			cdf_img_cpu[i] = (1.0f - MIN_PMF) * cdf_img_cpu[i] * norm + MIN_PMF * (float)(i+1) / (float)m_nerf.training.dataset.n_images;
		}
		m_nerf.training.error_map.cdf_img.copy_from_host(cdf_img_cpu);

		// Reset counters and decrease update rate.
		m_nerf.training.n_steps_since_error_map_update = 0;
		m_nerf.training.n_rays_since_error_map_update = 0;
		m_nerf.training.error_map.is_cdf_valid = true;

		m_nerf.training.n_steps_between_error_map_updates = (uint32_t)(m_nerf.training.n_steps_between_error_map_updates * 1.5f);
	}

	// Get extrinsics gradients
	m_nerf.training.n_steps_since_cam_update += n_training_steps;

	bool train_camera = m_nerf.training.optimize_extrinsics || m_nerf.training.optimize_distortion || m_nerf.training.optimize_focal_length || m_nerf.training.optimize_exposure;
	if (train_camera && m_nerf.training.n_steps_since_cam_update >= m_nerf.training.n_steps_between_cam_updates) {
		float per_camera_loss_scale = (float)m_nerf.training.n_images / LOSS_SCALE / (float)m_nerf.training.n_steps_between_cam_updates;

		if (m_nerf.training.optimize_extrinsics) {
			CUDA_CHECK_THROW(hipMemcpyAsync(m_nerf.training.cam_pos_gradient.data(), m_nerf.training.cam_pos_gradient_gpu.data(), m_nerf.training.cam_pos_gradient_gpu.get_bytes(), hipMemcpyDeviceToHost, stream));
			CUDA_CHECK_THROW(hipMemcpyAsync(m_nerf.training.cam_rot_gradient.data(), m_nerf.training.cam_rot_gradient_gpu.data(), m_nerf.training.cam_rot_gradient_gpu.get_bytes(), hipMemcpyDeviceToHost, stream));

			CUDA_CHECK_THROW(hipStreamSynchronize(stream));

			// Optimization step
			for (uint32_t i = 0; i < m_nerf.training.n_images; ++i) {
				Vector3f pos_gradient = m_nerf.training.cam_pos_gradient[i] * per_camera_loss_scale;
				Vector3f rot_gradient = m_nerf.training.cam_rot_gradient[i] * per_camera_loss_scale;

				float l2_reg = 0.01f;
				pos_gradient += m_nerf.training.cam_pos_offset[i].variable() * l2_reg;
				rot_gradient += m_nerf.training.cam_rot_offset[i].variable() * l2_reg;

				m_nerf.training.cam_pos_offset[i].set_learning_rate(std::max(1e-3f * std::pow(0.33f, (float)(m_training_step / 2048)), m_optimizer->learning_rate()/1000.0f));
				m_nerf.training.cam_rot_offset[i].set_learning_rate(std::max(1e-3f * std::pow(0.33f, (float)(m_training_step / 2048)), m_optimizer->learning_rate()/1000.0f));

				m_nerf.training.cam_pos_offset[i].step(pos_gradient);
				m_nerf.training.cam_rot_offset[i].step(rot_gradient);
			}

			update_nerf_transforms();
		}

		if (m_nerf.training.optimize_distortion) {
			linear_kernel(safe_divide, 0, stream,
				m_distortion.map->n_params(),
				m_distortion.map->gradients(),
				m_distortion.map->gradient_weights()
			);
			m_distortion.trainer->optimizer_step(stream, LOSS_SCALE*(float)m_nerf.training.n_steps_between_cam_updates);
		}

		if (m_nerf.training.optimize_focal_length) {
			CUDA_CHECK_THROW(hipMemcpyAsync(m_nerf.training.cam_focal_length_gradient.data(),m_nerf.training.cam_focal_length_gradient_gpu.data(),m_nerf.training.cam_focal_length_gradient_gpu.get_bytes(),hipMemcpyDeviceToHost, stream));
			CUDA_CHECK_THROW(hipStreamSynchronize(stream));
			Vector2f focal_length_gradient = m_nerf.training.cam_focal_length_gradient * per_camera_loss_scale;
			float l2_reg = 0.01f;
			focal_length_gradient += m_nerf.training.cam_focal_length_offset.variable() * l2_reg;
			m_nerf.training.cam_focal_length_offset.set_learning_rate(std::max(1e-3f * std::pow(0.33f, (float)(m_training_step / 2048)),m_optimizer->learning_rate() / 1000.0f));
			m_nerf.training.cam_focal_length_offset.step(focal_length_gradient);
			update_nerf_focal_lengths();
		}

		if (m_nerf.training.optimize_exposure) {
			CUDA_CHECK_THROW(hipMemcpyAsync(m_nerf.training.cam_exposure_gradient.data(), m_nerf.training.cam_exposure_gradient_gpu.data(), m_nerf.training.cam_exposure_gradient_gpu.get_bytes(), hipMemcpyDeviceToHost, stream));

			Array3f mean_exposure = Array3f::Constant(0.0f);

			// Optimization step
			for (uint32_t i = 0; i < m_nerf.training.n_images; ++i) {
				Array3f gradient = m_nerf.training.cam_exposure_gradient[i] * per_camera_loss_scale;

				float l2_reg = 0.00f;
				gradient += m_nerf.training.cam_exposure[i].variable() * l2_reg;

				m_nerf.training.cam_exposure[i].set_learning_rate(m_optimizer->learning_rate());
				m_nerf.training.cam_exposure[i].step(gradient);

				mean_exposure += m_nerf.training.cam_exposure[i].variable();
			}

			mean_exposure /= m_nerf.training.n_images;

			// Renormalize
			std::vector<Array3f> cam_exposures(m_nerf.training.n_images);
			for (uint32_t i = 0; i < m_nerf.training.n_images; ++i) {
				cam_exposures[i] = m_nerf.training.cam_exposure[i].variable() -= mean_exposure;
			}

			CUDA_CHECK_THROW(hipMemcpyAsync(m_nerf.training.cam_exposure_gpu.data(), cam_exposures.data(), m_nerf.training.cam_exposure_gpu.get_bytes(), hipMemcpyHostToDevice, stream));
		}

		m_nerf.training.n_steps_since_cam_update = 0;
	}
}

void Testbed::train_nerf_step(uint32_t target_batch_size, uint32_t n_rays_per_batch, uint32_t* counter, uint32_t* compacted_counter, float* loss, hipStream_t stream) {
	const uint32_t padded_output_width = m_network->padded_output_width();
	const uint32_t max_samples = target_batch_size * 16; // Somewhat of a worst case

	m_nerf.training.ray_counter.enlarge(1);

	GPUMemoryArena::Allocation alloc;
	auto scratch = allocate_workspace_and_distribute<
		uint32_t, // ray_indices
		Ray, // rays
		uint32_t, // numsteps
		NerfCoordinate, // coords
		float, // max_level
		network_precision_t, // mlp_out
		network_precision_t, // mlp_out_trimmed
		network_precision_t, // dloss_dmlp_out
		NerfCoordinate, // coords_compacted
		NerfCoordinate, // coords_gradient
		float // max_level_compacted
	>(
		stream, &alloc,
		n_rays_per_batch,
		n_rays_per_batch,
		n_rays_per_batch * 2,
		max_samples,
		max_samples,
		std::max(target_batch_size, max_samples) * padded_output_width,
		std::max(target_batch_size, max_samples) * 4,
		target_batch_size * padded_output_width,
		target_batch_size,
		target_batch_size,
		target_batch_size
	);

	// TODO: C++17 structured binding
	uint32_t* ray_indices = std::get<0>(scratch);
	Ray* rays = std::get<1>(scratch);
	uint32_t* numsteps = std::get<2>(scratch);
	NerfCoordinate* coords = std::get<3>(scratch);
	float* max_level = std::get<4>(scratch);
	network_precision_t* mlp_out = std::get<5>(scratch);
	network_precision_t* mlp_out_trimmed = std::get<6>(scratch);
	network_precision_t* dloss_dmlp_out = std::get<7>(scratch);
	NerfCoordinate* coords_compacted = std::get<8>(scratch);
	NerfCoordinate* coords_gradient = std::get<9>(scratch);
	float* max_level_compacted = std::get<10>(scratch);

	uint32_t max_inference;
	if (m_nerf.training.measured_batch_size_before_compaction == 0) {
		m_nerf.training.measured_batch_size_before_compaction = max_inference = max_samples;
	} else {
		max_inference = next_multiple(std::min(m_nerf.training.measured_batch_size_before_compaction, max_samples), BATCH_SIZE_MULTIPLE);
	}

	GPUMatrix<float> coords_matrix((float*)coords, sizeof(NerfCoordinate)/sizeof(float), max_inference);
	GPUMatrix<network_precision_t> rgbsigma_matrix(mlp_out, padded_output_width, max_inference);

	GPUMatrix<float> compacted_coords_matrix((float*)coords_compacted, sizeof(NerfCoordinate)/sizeof(float), target_batch_size);
	GPUMatrix<network_precision_t> compacted_rgbsigma_matrix(mlp_out, padded_output_width, target_batch_size);

	GPUMatrix<network_precision_t> gradient_matrix(dloss_dmlp_out, padded_output_width, target_batch_size);

	if (m_training_step == 0) {
		m_nerf.training.n_rays_total = 0;
	}

	uint32_t n_rays_total = m_nerf.training.n_rays_total;
	m_nerf.training.n_rays_total += n_rays_per_batch;
	m_nerf.training.n_rays_since_error_map_update += n_rays_per_batch;

	// If we have an envmap, prepare its gradient buffer
	float* envmap_gradient = m_nerf.training.train_envmap ? m_envmap.envmap->gradients() : nullptr;

	bool sample_focal_plane_proportional_to_error = m_nerf.training.error_map.is_cdf_valid && m_nerf.training.sample_focal_plane_proportional_to_error;
	bool sample_image_proportional_to_error = m_nerf.training.error_map.is_cdf_valid && m_nerf.training.sample_image_proportional_to_error;
	bool include_sharpness_in_error = m_nerf.training.include_sharpness_in_error;
	// This is low-overhead enough to warrant always being on.
	// It makes for useful visualizations of the training error.
	bool accumulate_error = true;

	CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.ray_counter.data(), 0, sizeof(uint32_t), stream));

	linear_kernel(generate_training_samples_nerf, 0, stream,
		n_rays_per_batch,
		m_aabb,
		max_inference,
		n_rays_total,
		m_rng,
		m_nerf.training.dataset.rays_data.data(),
		m_nerf.training.ray_counter.data(),
		counter,
		ray_indices,
		rays,
		numsteps,
		coords,
		m_nerf.training.image_resolution,
		m_nerf.training.n_images,
		m_nerf.training.dataset.principal_point,
		m_nerf.training.focal_lengths_gpu.data(),
		m_nerf.training.transforms_gpu.data(),
		m_nerf.training.dataset.camera_distortion,
		m_nerf.density_grid_bitfield.data(),
		m_max_level_rand_training,
		max_level,
		m_nerf.training.snap_to_pixel_centers,
		m_nerf.training.train_envmap,
		m_nerf.cone_angle_constant,
		m_distortion.map->params(),
		m_distortion.resolution,
		sample_focal_plane_proportional_to_error ? m_nerf.training.error_map.cdf_x_cond_y.data() : nullptr,
		sample_focal_plane_proportional_to_error ? m_nerf.training.error_map.cdf_y.data() : nullptr,
		sample_image_proportional_to_error ? m_nerf.training.error_map.cdf_img.data() : nullptr,
		m_nerf.training.error_map.cdf_resolution,
		m_nerf.training.near_distance,
		m_nerf.training.dataset.images_data.data()
	);

	auto hg_enc = dynamic_cast<GridEncoding<network_precision_t>*>(m_encoding.get());
	if (hg_enc) {
		hg_enc->set_max_level_gpu(m_max_level_rand_training ? max_level : nullptr);
	}

	m_network->inference_mixed_precision(stream, coords_matrix, rgbsigma_matrix, false);
	if (padded_output_width != 4) {
		linear_kernel(tcnn::trim<network_precision_t>, 0, stream, max_inference*4, padded_output_width, 4, mlp_out, mlp_out_trimmed);
		mlp_out = mlp_out_trimmed;
	}

	if (hg_enc) {
		hg_enc->set_max_level_gpu(m_max_level_rand_training ? max_level_compacted : nullptr);
	}

	linear_kernel(compute_loss_kernel_train_nerf, 0, stream,
		n_rays_per_batch,
		m_aabb,
		n_rays_total,
		m_rng,
		target_batch_size,
		m_nerf.training.ray_counter.data(),
		LOSS_SCALE,
		padded_output_width,
		m_envmap.envmap->params(),
		envmap_gradient,
		m_envmap.resolution,
		m_envmap.loss_type,
		m_background_color.head<3>(),
		m_color_space,
		m_nerf.training.random_bg_color,
		m_nerf.training.linear_colors,
		m_nerf.training.dataset.images_data.data(),
		m_nerf.training.n_images,
		m_nerf.training.image_resolution,
		mlp_out,
		compacted_counter,
		ray_indices,
		rays,
		numsteps,
		coords,
		coords_compacted,
		dloss_dmlp_out,
		m_nerf.training.loss_type,
		loss,
		m_max_level_rand_training,
		max_level_compacted,
		m_nerf.rgb_activation,
		m_nerf.density_activation,
		m_nerf.training.snap_to_pixel_centers,
		accumulate_error ? m_nerf.training.error_map.data.data() : nullptr,
		sample_focal_plane_proportional_to_error ? m_nerf.training.error_map.cdf_x_cond_y.data() : nullptr,
		sample_focal_plane_proportional_to_error ? m_nerf.training.error_map.cdf_y.data() : nullptr,
		sample_image_proportional_to_error ? m_nerf.training.error_map.cdf_img.data() : nullptr,
		m_nerf.training.error_map.resolution,
		m_nerf.training.error_map.cdf_resolution,
		include_sharpness_in_error ? m_nerf.training.dataset.sharpness_data.data() : nullptr,
		m_nerf.training.dataset.sharpness_resolution,
		m_nerf.training.sharpness_grid.data(),
		m_nerf.density_grid.data(),
		m_nerf.density_grid_mean.data(),
		m_nerf.training.cam_exposure_gpu.data(),
		m_nerf.training.optimize_exposure ? m_nerf.training.cam_exposure_gradient_gpu.data() : nullptr
	);

	fill_rollover_and_rescale<network_precision_t><<<n_blocks_linear(target_batch_size*padded_output_width), n_threads_linear, 0, stream>>>(
		target_batch_size, padded_output_width, compacted_counter, dloss_dmlp_out
	);
	fill_rollover<NerfCoordinate><<<n_blocks_linear(target_batch_size), n_threads_linear, 0, stream>>>(
		target_batch_size, 1, compacted_counter, coords_compacted
	);
	fill_rollover<float><<<n_blocks_linear(target_batch_size), n_threads_linear, 0, stream>>>(
		target_batch_size, 1, compacted_counter, max_level_compacted
	);

	bool train_camera = m_nerf.training.optimize_extrinsics || m_nerf.training.optimize_distortion || m_nerf.training.optimize_focal_length;
	m_network->forward(stream, compacted_coords_matrix, &compacted_rgbsigma_matrix, false, train_camera);

	GPUMatrix<float> coords_gradient_matrix((float*)coords_gradient, sizeof(NerfCoordinate)/sizeof(float), target_batch_size);
	m_network->backward(stream, compacted_coords_matrix, compacted_rgbsigma_matrix, gradient_matrix, train_camera ? &coords_gradient_matrix : nullptr);

	if (train_camera) {
		// Compute camera gradients
		linear_kernel(compute_cam_gradient_train_nerf, 0, stream,
			n_rays_per_batch,
			n_rays_total,
			m_rng,
			m_aabb,
			m_nerf.training.ray_counter.data(),
			m_nerf.training.transforms_gpu.data(),
			m_nerf.training.image_resolution,
			m_nerf.training.snap_to_pixel_centers,
			m_nerf.training.optimize_extrinsics ? m_nerf.training.cam_pos_gradient_gpu.data() : nullptr,
			m_nerf.training.optimize_extrinsics ? m_nerf.training.cam_rot_gradient_gpu.data() : nullptr,
			m_nerf.training.n_images,
			ray_indices,
			rays,
			numsteps,
			coords_compacted,
			coords_gradient,
			m_nerf.training.optimize_distortion ? m_distortion.map->gradients() : nullptr,
			m_nerf.training.optimize_distortion ? m_distortion.map->gradient_weights() : nullptr,
			m_distortion.resolution,
			m_nerf.training.optimize_focal_length ? m_nerf.training.cam_focal_length_gradient_gpu.data() : nullptr,
			sample_focal_plane_proportional_to_error ? m_nerf.training.error_map.cdf_x_cond_y.data() : nullptr,
			sample_focal_plane_proportional_to_error ? m_nerf.training.error_map.cdf_y.data() : nullptr,
			sample_image_proportional_to_error ? m_nerf.training.error_map.cdf_img.data() : nullptr,
			m_nerf.training.error_map.cdf_resolution
		);
	}

	m_rng.advance();
	m_trainer->optimizer_step(stream, LOSS_SCALE);

	++m_training_step;

	if (hg_enc) {
		hg_enc->set_max_level_gpu(nullptr);
	}
}

void Testbed::training_prep_nerf(uint32_t batch_size, uint32_t n_training_steps, hipStream_t stream) {
	float alpha = std::pow(m_nerf.training.density_grid_decay, n_training_steps / 16.0f);
	uint32_t n_cascades = m_nerf.max_cascade+1;
	if (m_training_step < 256) {
		update_density_grid_nerf(alpha, NERF_GRIDSIZE()*NERF_GRIDSIZE()*NERF_GRIDSIZE()*n_cascades, 0, stream);
	} else {
		update_density_grid_nerf(alpha, NERF_GRIDSIZE()*NERF_GRIDSIZE()*NERF_GRIDSIZE()/4*n_cascades, NERF_GRIDSIZE()*NERF_GRIDSIZE()*NERF_GRIDSIZE()/4*n_cascades, stream);
	}
}

void Testbed::optimise_mesh_step(uint32_t N_STEPS) {
	uint32_t n_verts = (uint32_t)m_mesh.verts.size();
	if (!n_verts) {
		return;
	}

	const uint32_t padded_output_width = m_nerf_network->padded_density_output_width();
	GPUMemory<NerfCoordinate> coords(n_verts);
	GPUMemory<network_precision_t> mlp_out(n_verts * padded_output_width);

	GPUMatrix<float> positions_matrix((float*)coords.data(), sizeof(NerfCoordinate)/sizeof(float), n_verts);
	GPUMatrix<network_precision_t> density_matrix(mlp_out.data(), padded_output_width, n_verts);

	for (uint32_t i = 0; i < N_STEPS; ++i) {
		linear_kernel(generate_nerf_network_inputs_from_positions, 0, m_inference_stream, n_verts, m_aabb, m_mesh.verts.data(), coords.data());

		// For each optimizer step, we need the density at the given pos...
		m_nerf_network->density(m_inference_stream, positions_matrix, density_matrix);
		// ...as well as the input gradient w.r.t. density, which we will store in the nerf coords.
		m_nerf_network->input_gradient(m_inference_stream, 3, positions_matrix, positions_matrix);
		// and the 1ring centroid for laplacian smoothing
		compute_mesh_1ring(m_mesh.verts, m_mesh.indices, m_mesh.verts_smoothed, m_mesh.vert_normals);

		// With these, we can compute a gradient that points towards the threshold-crossing of density...
		compute_mesh_opt_gradients(m_mesh.thresh, m_mesh.verts, m_mesh.vert_normals, m_mesh.verts_smoothed, padded_output_width, mlp_out.data(), sizeof(NerfCoordinate)/sizeof(float), (const float*)coords.data(), m_mesh.verts_gradient, m_mesh.smooth_amount, m_mesh.density_amount, m_mesh.inflate_amount);

		// ...that we can pass to the optimizer.
		m_mesh.verts_optimizer->step(m_inference_stream, 1.0f, (float*)m_mesh.verts.data(), (float*)m_mesh.verts.data(), (float*)m_mesh.verts_gradient.data());
	}
}

void Testbed::compute_mesh_vertex_colors() {
	uint32_t n_verts = (uint32_t)m_mesh.verts.size();
	if (!n_verts) {
		return;
	}

	m_mesh.vert_colors.resize(n_verts);
	const uint32_t padded_output_width = m_network->padded_output_width();
	GPUMemory<NerfCoordinate> coords(n_verts);
	GPUMemory<float> mlp_out(n_verts* padded_output_width);

	GPUMatrix<float> positions_matrix((float*)coords.data(), sizeof(NerfCoordinate)/sizeof(float), n_verts);
	GPUMatrix<float> color_matrix(mlp_out.data(), padded_output_width, n_verts);
	linear_kernel(generate_nerf_network_inputs_from_positions, 0, m_inference_stream, n_verts, m_aabb, m_mesh.verts.data(), coords.data());
	m_network->inference(m_inference_stream, positions_matrix, color_matrix);
	linear_kernel(extract_srgb_with_activation, 0, m_inference_stream, n_verts*3, 3, mlp_out.data(), (float*)m_mesh.vert_colors.data(), m_nerf.rgb_activation, m_nerf.training.linear_colors);
}

GPUMemory<float> Testbed::get_density_on_grid(Vector3i res3d, const BoundingBox& aabb) {
	const uint32_t n_elements = (res3d.x()*res3d.y()*res3d.z());
	GPUMemory<float> density(n_elements);

	const uint32_t batch_size = std::min(n_elements, 1u<<20);
	bool nerf_mode = m_testbed_mode == ETestbedMode::Nerf;

	const uint32_t padded_output_width = nerf_mode ? m_nerf_network->padded_density_output_width() : m_network->padded_output_width();

	GPUMemoryArena::Allocation alloc;
	auto scratch = allocate_workspace_and_distribute<
		NerfPosition,
		network_precision_t
	>(m_inference_stream, &alloc, n_elements, batch_size * padded_output_width);

	NerfPosition* positions = std::get<0>(scratch);
	network_precision_t* mlp_out = std::get<1>(scratch);

	const dim3 threads = { 16, 8, 1 };
	const dim3 blocks = { div_round_up((uint32_t)res3d.x(), threads.x), div_round_up((uint32_t)res3d.y(), threads.y), div_round_up((uint32_t)res3d.z(), threads.z) };

	BoundingBox unit_cube = BoundingBox{Vector3f::Zero(), Vector3f::Ones()};
	generate_grid_samples_nerf_uniform<<<blocks, threads, 0, m_inference_stream>>>(res3d, m_nerf.density_grid_ema_step, aabb, nerf_mode ? m_aabb : unit_cube , positions);

	// Only process 1m elements at a time
	for (uint32_t offset = 0; offset < n_elements; offset += batch_size) {
		uint32_t local_batch_size = std::min(n_elements - offset, batch_size);

		GPUMatrix<network_precision_t> density_matrix(mlp_out, padded_output_width, local_batch_size);
		if (nerf_mode)
			m_nerf_network->density(m_inference_stream, {(float*)(positions + offset), sizeof(NerfPosition)/sizeof(float)}, density_matrix);
		else {
			GPUMatrix<float> positions_matrix((float*)(positions + offset), sizeof(NerfPosition)/sizeof(float), local_batch_size);
			m_network->inference_mixed_precision(m_inference_stream, positions_matrix, density_matrix);
		}
		linear_kernel(grid_samples_half_to_float, 0, m_inference_stream,
			local_batch_size,
			m_aabb,
			density.data() + offset , //+ axis_step * n_elements,
			padded_output_width,
			mlp_out,
			m_nerf.density_activation,
			positions + offset,
			nerf_mode ? m_nerf.density_grid.data() : nullptr
		);
	}

	return density;
}

GPUMemory<Eigen::Array4f> Testbed::get_rgba_on_grid(Vector3i res3d, Eigen::Vector3f ray_dir) {
	const uint32_t n_elements = (res3d.x()*res3d.y()*res3d.z());
	GPUMemory<Eigen::Array4f> rgba(n_elements);
	GPUMemory<NerfCoordinate> positions(n_elements);
	const uint32_t batch_size = std::min(n_elements, 1u<<20);

	// generate inputs
	const dim3 threads = { 16, 8, 1 };
	const dim3 blocks = { div_round_up((uint32_t)res3d.x(), threads.x), div_round_up((uint32_t)res3d.y(), threads.y), div_round_up((uint32_t)res3d.z(), threads.z) };
	generate_grid_samples_nerf_uniform_dir<<<blocks, threads, 0, m_inference_stream>>>(res3d, m_nerf.density_grid_ema_step, m_render_aabb, m_aabb, ray_dir, positions.data());

	// Only process 1m elements at a time
	for (uint32_t offset = 0; offset < n_elements; offset += batch_size) {
		uint32_t local_batch_size = std::min(n_elements - offset, batch_size);

		// run network
		GPUMatrix<float> positions_matrix((float*) (positions.data() + offset), sizeof(NerfCoordinate)/sizeof(float), local_batch_size);
		GPUMatrix<float> rgbsigma_matrix((float*) (rgba.data() + offset), 4, local_batch_size);
		m_network->inference(m_inference_stream, positions_matrix, rgbsigma_matrix);

		// convert network output to RGBA (in place)
		linear_kernel(compute_nerf_density, 0, m_inference_stream, local_batch_size, rgba.data() + offset, m_nerf.rgb_activation, m_nerf.density_activation);
	}
	return rgba;
}

int Testbed::marching_cubes(Vector3i res3d, const BoundingBox& aabb, float thresh) {
	res3d.x() = next_multiple((unsigned int)res3d.x(), 16u);
	res3d.y() = next_multiple((unsigned int)res3d.y(), 16u);
	res3d.z() = next_multiple((unsigned int)res3d.z(), 16u);

	GPUMemory<float> density = get_density_on_grid(res3d, aabb);
	marching_cubes_gpu(m_inference_stream, m_render_aabb, res3d, thresh, density, m_mesh.verts, m_mesh.indices);

	uint32_t n_verts = (uint32_t)m_mesh.verts.size();
	m_mesh.verts_gradient.resize(n_verts);

	m_mesh.trainable_verts = std::make_shared<TrainableBuffer<3, 1, float>>(Matrix<int, 1, 1>{(int)n_verts});
	m_mesh.verts_gradient.copy_from_device(m_mesh.verts); // Make sure the vertices don't get destroyed in the initialization

	pcg32 rnd{m_seed};
	m_mesh.trainable_verts->initialize_params(rnd, (float*)m_mesh.verts.data(), (float*)m_mesh.verts.data(), (float*)m_mesh.verts.data(), (float*)m_mesh.verts.data(), (float*)m_mesh.verts_gradient.data());
	m_mesh.verts.copy_from_device(m_mesh.verts_gradient);

	m_mesh.verts_optimizer.reset(create_optimizer<float>({
		{"otype", "Adam"},
		{"learning_rate", 1e-4},
		{"beta1", 0.9f},
		{"beta2", 0.99f},
	}));

	m_mesh.verts_optimizer->allocate(m_mesh.trainable_verts);

	compute_mesh_1ring(m_mesh.verts, m_mesh.indices, m_mesh.verts_smoothed, m_mesh.vert_normals);
	compute_mesh_vertex_colors();
	return (int)(m_mesh.indices.size()/3);
}

uint8_t* Testbed::Nerf::get_density_grid_bitfield_mip(uint32_t mip) {
	return density_grid_bitfield.data() + grid_mip_offset(mip)/8;
}

int Testbed::find_best_training_view(int default_view) {
	int bestimage=default_view;
	float bestscore=1000.f;
	for (int i=0;i<m_nerf.training.dataset.n_images;++i) {
		float score=(m_nerf.training.dataset.xforms[i].col(3)-m_camera.col(3)).norm();
		score+=0.25f * (m_nerf.training.dataset.xforms[i].col(2)-m_camera.col(2)).norm();
		if (score<bestscore) { bestscore=score; bestimage=i; }
	}
	return bestimage;
}

NGP_NAMESPACE_END
